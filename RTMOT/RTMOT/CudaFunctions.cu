#include "hip/hip_runtime.h"
 // Copyright 2012 by Thanuja Mallikarachchi, Chammi Dilhari,  Chathurangi Kumarasinghe, Tharindu Bandaragoda
 //
 // This file is part of RTMOT.
 //
 // TLDCUDA is free software: you can redistribute it and/or modify
 // it under the terms of the GNU General Public License as published by
 // the Free Software Foundation, either version 3 of the License, or
 // (at your option) any later version.
 //
 // TLDCUDA is distributed in the hope that it will be useful,
 // but WITHOUT ANY WARRANTY; without even the implied warranty of
 // MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 // GNU General Public License for more details.
 //
 // You should have received a copy of the GNU General Public License
 // along with TLDCUDA.  If not, see <http://www.gnu.org/licenses/>.

#include "stdafx.h"
#include "CudaFunctions.h"
#define patchSize 45
#define steps 15
#define SmoothYNoOfThreads 1
#define BlurNoOfThreads 1

//scales for each pyramid 
const float pyramidScales[] = {1, 0.5f, 0.25f, 0.125f, 0.0625f, 0.03125f, 0.015625f, 0.0078125f};

texture<float, 2, hipReadModeElementType> texturePreviousFrame;
texture<float, 2, hipReadModeElementType> textureCurrentFrame;
texture<float, 2, hipReadModeElementType> texturePyramid;

texture<unsigned char, 2, hipReadModeElementType> textureImage;
texture<unsigned char, 2, hipReadModeElementType> textureBlurredImage;
texture<unsigned char, 2, hipReadModeElementType> textureWarpedImage;
texture<float, 2, hipReadModeElementType> textureIntregalImage;
texture<float, 2, hipReadModeElementType> textureIntregalSQImage;
texture<unsigned char, 2, hipReadModeElementType> textureRowBlurredImage;

//-------------------n tuples ------------------kernel//
__global__ void calc_grid_kernel(int* x1,int* x2,int* x3,int x1_cols,int x2_cols)
{
	int idx_x = blockDim.x*blockIdx.x + threadIdx.x;
	int numDim=x1_cols*x2_cols;
	int idx = numDim*idx_x;

	int idx_x1_col =(int)floorf(idx_x/x2_cols);
	int idx_x2_col = idx_x-idx_x1_col*x2_cols;

	x3[idx_x]=x1[idx_x1_col];
	x3[x1_cols*x2_cols+idx_x]=x2[idx_x2_col];
}


//////////////-------------------gpu IntregalImage and Square Intregal Image Generation -----------------------------------/////////////////////
/*Intregal Image Calculation is Implementation of paper "Efficient Integral Image Computation on the GPU" see the paper for clarifications*/
__global__ void rowIntegralKernel(float *gpu_rowImage,int w,int h) {

  const int rowSize = 1024;		
__shared__ float input[rowSize];
	int idxN = threadIdx.x;
	int idx = 2*threadIdx.x;

	if (idx <w){
		input[idx] = tex2D(textureImage,(float)idx,(float)blockIdx.x);
		input[idx+1] = tex2D(textureImage,(float)(idx+1),(float)blockIdx.x);
	}
	else {
		input[idx] = 0.0f;
		input[idx+1] = 0.0f;
	}
		
	
	int offset = 1;
	
	for(int d = rowSize>>1; d > 0; d >>= 1){
			
		__syncthreads();
				
		if(idxN < d){
			int ai = offset*(idx+1)-1;
			int bi = offset*(idx+2)-1;
			input[bi] += input[ai];
		}
		offset *= 2;
	}
		
	if(idxN == 0) input[rowSize - 1] = 0.0f;
	
	for(int d = 1; d < rowSize; d *= 2){
		offset >>= 1; __syncthreads();
			if(idxN < d){
				int ai = offset*(idx+1)-1;
				int bi = offset*(idx+2)-1;
				float t = input[ai];
				input[ai] = input[bi];
				input[bi] += t;
			}
	}
	__syncthreads();

	if((idx+1) < w){
		gpu_rowImage[blockIdx.x*w+idx]=input[idx+1];
		gpu_rowImage[blockIdx.x*w+idx+1]=input[idx+2];
	}
}


__global__ void colIntegralKernel(float *gpu_rowImage,float *gpu_intregalImage,int w,int h,int pitch) {
	const int colSize = 512;	
	__shared__ float input[colSize];
	int idxN = threadIdx.x;
	int idx = 2*threadIdx.x;

	if (idx<h){
		input[idx] = gpu_rowImage[idx*w + blockIdx.x];
		input[idx+1] = gpu_rowImage[(idx+1)*w + blockIdx.x];
	}
	else {
	input[idx] = 0.0f;
	input[idx+1] = 0.0f;
	}
		
	
	int offset = 1;
	
	for(int d = colSize>>1; d > 0; d >>= 1){
			
		__syncthreads();
				
		if(idxN < d){
			int ai = offset*(idx+1)-1;
			int bi = offset*(idx+2)-1;
			input[bi] += input[ai];
		}
		offset *= 2;
	}
		
	if(idxN == 0) input[colSize - 1] = 0.0f;
	
	for(int d = 1; d < colSize; d *= 2){
		offset >>= 1; __syncthreads();
			if(idxN < d){
				int ai = offset*(idx+1)-1;
				int bi = offset*(idx+2)-1;
				float t = input[ai];
				input[ai] = input[bi];
				input[bi] += t;
			}
	}
	__syncthreads();

	if((idx+1) < h){
		gpu_intregalImage[blockIdx.x + idx*pitch]=input[idx+1];
		gpu_intregalImage[blockIdx.x + (idx+1)*pitch]=input[idx+2];
	}
}

__global__ void rowIntegralSQKernel(float *gpu_rowSQImage,int w,int h) {

	const int rowSize = 1024;		
	__shared__ float input[rowSize];
	int idxN = threadIdx.x;
	int idx = 2*threadIdx.x;

	if (idx <w){
		float i = tex2D(textureImage,(float)idx,(float)blockIdx.x);
		float j = tex2D(textureImage,(float)(idx+1),(float)blockIdx.x);
		input[idx] = i * i;
		input[idx+1] = j * j;
	}
	else {
		input[idx] = 0.0f;
		input[idx+1] = 0.0f;
	}
		
	int offset = 1;
	
	for(int d = rowSize>>1; d > 0; d >>= 1){
			
		__syncthreads();
				
		if(idxN < d){
			int ai = offset*(idx+1)-1;
			int bi = offset*(idx+2)-1;
			input[bi] += input[ai];
		}
		offset *= 2;
	}
		
	if(idxN == 0) input[rowSize - 1] = 0.0f;
	
	for(int d = 1; d < rowSize; d *= 2){
		offset >>= 1; __syncthreads();
			if(idxN < d){
				int ai = offset*(idx+1)-1;
				int bi = offset*(idx+2)-1;
				float t = input[ai];
				input[ai] = input[bi];
				input[bi] += t;
			}
	}
	__syncthreads();

	if((idx+1) < w){
		gpu_rowSQImage[blockIdx.x*w+idx]=input[idx+1];
		gpu_rowSQImage[blockIdx.x*w+idx+1]=input[idx+2];
	}
}


__global__ void colIntegralSQKernel(float *gpu_rowSQImage,float *gpu_intregalSQImage,int w,int h,int pitch) {
	const int colSize = 512;	
__shared__ float input[colSize];
	int idxN = threadIdx.x;
	int idx = 2*threadIdx.x;

	if (idx<h){
		input[idx] = gpu_rowSQImage[idx*w + blockIdx.x];
		input[idx+1] = gpu_rowSQImage[(idx+1)*w + blockIdx.x];		
	}
	else {
	input[idx] = 0.0f;
	input[idx+1] = 0.0f;
	}
		
	
	int offset = 1;
	
	for(int d = colSize>>1; d > 0; d >>= 1){
			
		__syncthreads();
				
		if(idxN < d){
			int ai = offset*(idx+1)-1;
			int bi = offset*(idx+2)-1;
			input[bi] += input[ai];
		}
		offset *= 2;
	}
		
	if(idxN == 0) input[colSize - 1] = 0.0f;
	
	for(int d = 1; d < colSize; d *= 2){
		offset >>= 1; __syncthreads();
			if(idxN < d){
				int ai = offset*(idx+1)-1;
				int bi = offset*(idx+2)-1;
				float t = input[ai];
				input[ai] = input[bi];
				input[bi] += t;
			}
	}
	__syncthreads();

	if((idx+1) < h){
		gpu_intregalSQImage[blockIdx.x + idx*pitch]=input[idx+1];
		gpu_intregalSQImage[blockIdx.x + (idx+1)*pitch]=input[idx+2];
	}
}

///////////// --------------------GETPATTERN-------------------------------------/////////////////////
__global__ void floatToCharKernal(float* imageIn, unsigned char* imageOut,int height,int width, int imageOut_pitch){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if((x < width) && (y < height)) imageOut[y*imageOut_pitch + x] = imageIn[y*width + x];
}
__global__ void getPatternKernel( float* patchout,float* bb) {	
	
__shared__ float patch[patchSize][patchSize];
	__shared__ float total[steps][steps];
	__shared__ float rowSum[steps];
	__shared__ float sum;
	
	int boxWidth,boxHeight;
	int boxX,boxY;
	total[threadIdx.y][threadIdx.x]=0.0f;
	boxX = bb[4*blockIdx.x+0];
	boxY = bb[4*blockIdx.x+1];
	boxWidth	= bb[4*blockIdx.x+2] - boxX;
	boxHeight	= bb[4*blockIdx.x+3] - boxY;
	int threadw=patchSize/steps;
		
	int x0,y0;	
	float px,py;
	float p0,p1,p2,p3;
	float px0,py0;
	float dx,dy;

	for(int i=0;i<threadw;i++){
		y0=threadIdx.y*threadw+i;
		py = (float)boxY + (float)y0*((float)boxHeight/(float)patchSize);
		
			for(int j=0;j<threadw;j++){
				x0= threadIdx.x*threadw+j;
				px =(float) boxX + (float)x0*((float)boxWidth/(float)patchSize);

				px0=floorf(px);
				py0=floorf(py);				
				  dx=px-px0;
				  dy=py-py0;

				 p0 = tex2D(textureWarpedImage,px0,py0);
				 p1 = tex2D(textureWarpedImage,px0+1.0f,py0);
				 p2 = tex2D(textureWarpedImage,px0+1.0f,py0+1.0f);
				 p3 = tex2D(textureWarpedImage,px0,py0+1.0f);
 	
				 patch[y0][x0]=(p0*(1.0f-dx)+p1*dx)*(1.0f-dy)+(p3*(1.0f-dx)+p2*dx)*dy;				
				 total[threadIdx.y][threadIdx.x]  += patch[y0][x0];
			}
	}
	
	__syncthreads();
		
	if(threadIdx.x==0){
		rowSum[threadIdx.y]=0.0f;
		
		for(int i=0;i<steps;i++){
			rowSum[threadIdx.y] += total[threadIdx.y][i];
		}
	}
	
	__syncthreads();

    if(threadIdx.x==0 && threadIdx.y==0){
		sum=0.0f;
		
		for(int i=0;i<steps;i++){
			sum += rowSum[i];
		}
	}
   	__syncthreads();
	sum /= (float)(patchSize*patchSize);


	for(int i=0;i<threadw;i++){
		y0=threadIdx.y*threadw+i;
        
		for(int j=0;j<threadw;j++){
			x0=threadIdx.x*threadw+j;
			patchout[blockIdx.x*2025+y0*patchSize+x0]= patch[y0][x0] - sum;
			}
		}
}

///////////// -------------------- TRACKER-------------------------------------/////////////////////

/*Calculate Normalized Crosscorelation between given data sets: for the calculation a 13 by 13 pixel window is taken around each pixel for the crosscorelation calculation*/
__global__ void find_ncc_kernel(int nPts,int Winsize,float* d_Ipts,float* d_Jpts,char* d_status,float* d_ncc,int w,int h)
{

	__shared__ float ixy[64][96];
	__shared__ float jxy[64][96];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx > nPts-1)
		return;

	int iSum = 0;
	int jSum = 0;
	
	float ix = d_Ipts[2*idx  ];
	float iy = d_Ipts[2*idx+1];
	float jx = d_Jpts[2*idx  ];
	float jy = d_Jpts[2*idx+1];
	int halfWinSize  = (Winsize-1)*0.5f;

	float jNew = 0.0f;
	float iNew = 0.0f;
	
	float numerator = 0.0f;
	float denominator = 0.0f;

	float jsqSum = 0.0f;
	float isqSum = 0.0f;

	if(d_status[idx] == 1) //corelation is calculated only if there is a valid optical flow calculation for that point
	{
		// get the summation of all points inside the window
		for(int y = -halfWinSize ; y <= halfWinSize ; y++)
		{
			for(int x = -halfWinSize ; x <= halfWinSize ; x++)
			{
				ixy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize] = tex2D(textureCurrentFrame, ix + x, iy + y);
				jxy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize] = tex2D(texturePreviousFrame, jx + x, jy + y);				
				iSum += ixy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize];
				jSum += jxy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize];

			}
		}


		// find T' and I' and 
		iSum = iSum/(float)(w*h);
		jSum = jSum/(float)(w*h);
		// find T' and I' and 

		for(int x = -halfWinSize ; x<= halfWinSize ; x++)
		{
			for(int y = -halfWinSize ; y <= halfWinSize ; y++)
			{			
				iNew = ixy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize] - iSum; 

				jNew = jxy[threadIdx.x][(y+halfWinSize)*Winsize + x+halfWinSize] - jSum;

				numerator += (iNew * jNew); //find numerator

				isqSum += iNew*iNew;
				jsqSum += jNew*jNew;
			}
		}

		// find denominator
		float val = isqSum*jsqSum;
		denominator = sqrtf(val);


		d_ncc[idx] = (float)(numerator / denominator); //normalized crosscorelation
		
	}
	else
	{
		d_ncc[idx] = 0.0f; //if status is zero no point of calculating crosscorelation
	}
	__syncthreads();

}

/*Calculate Euclidian distance between 2 datasets*/
__global__ void calEuDistance(const int nPts,const float* ptsI,const float* ptsJ,float* distances){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx > nPts-1)
		return;

	float x = (ptsJ[2*idx]-ptsI[2*idx])*(ptsJ[2*idx]-ptsI[2*idx]);
	float y = (ptsJ[2*idx+1]-ptsI[2*idx+1])*(ptsJ[2*idx+1]-ptsI[2*idx+1]);
	distances[idx]  = sqrtf(x+y);
}


/*Convert image from RGB to Greay*/
__global__ void convertToGrey(unsigned char *d_in, float *d_out, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < N) 
        d_out[idx] = d_in[idx*3]*0.1144f + d_in[idx*3+1]*0.5867f + d_in[idx*3+2]*0.2989f;
}

/*Downsampling the image to 0.5 size*/
__global__ void pyrDownsample(int w1, int h1, float *out, int w2, int h2)
{
    // Input has to be greyscale
    int x2 = blockIdx.x*blockDim.x + threadIdx.x;
    int y2 = blockIdx.y*blockDim.y + threadIdx.y;

    if( (x2 < w2) && (y2 < h2) ) {    
        int x = x2*2;
        int y = y2*2;
		
		float val = 0.25f*tex2D(texturePyramid,x,y);
			  val += 0.125f*tex2D(texturePyramid,x-1,y);
			  val += 0.125f*tex2D(texturePyramid,x-2,y);
			  val += 0.125f*tex2D(texturePyramid,x,y-1);
			  val += 0.125f*tex2D(texturePyramid,x,y-2);
			  val += 0.0625f*tex2D(texturePyramid,x-1,y-1);
			  val += 0.0625f*tex2D(texturePyramid,x-1,y-2);
			  val += 0.0625f*tex2D(texturePyramid,x-2,y-1);
			  val += 0.0625f*tex2D(texturePyramid,x-2,y-2);
        out[y2*w2 + x2] = val;
    }
} 

/*Smoothing row wise*/
__global__ void smoothX( float* imageout,int width,int height) {	

	const int Blocksize=252; 
	int x = blockIdx.x*Blocksize+threadIdx.x;
	int pX = threadIdx.x;
	int y = blockIdx.y;
	
	__shared__ unsigned char imageBlock[Blocksize+4];	
	 
	imageBlock[threadIdx.x]= tex2D(texturePyramid,(x-2),y);
	__syncthreads();

	if( x >= width) return;
	
	if( pX < 252){
		imageout[ y*width + x]= 
		0.0625f*imageBlock[pX++]+
		0.25f*imageBlock[pX++]+
		0.375f*imageBlock[pX++]+
		0.25f*imageBlock[pX++]+
		0.0625f*imageBlock[pX];	
		}		
}

/*Smoothing column wise*/
__global__ void smoothY( float* imageout,int width,int height) {

	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y =  blockIdx.y;
	
	if((x < width) && (y < height)){

		for(int i=0; i<SmoothYNoOfThreads; i++){
			int yIndex = SmoothYNoOfThreads*y+i-2;		 
			float val = 0.0625f*tex2D(texturePyramid,x,yIndex++);
			val += 0.25f  *tex2D(texturePyramid,x,yIndex++);
			val += 0.375f *tex2D(texturePyramid,x,yIndex++);
			val += 0.25f  *tex2D(texturePyramid,x,yIndex++);
			val += 0.0625f*tex2D(texturePyramid,x,yIndex);
			imageout[yIndex * width + x ] = val;
		}
	}		
}


/*Calculate Optical Flow*/
__global__ void track(const int nBoxes,const int width, const int scaledHeights, 
                      float pyramidScales, int level, char initialGuess, 
                      float *ptsJ, char *status,float* ptsI)
{  
	__shared__ float Ix[256];
	__shared__ float Iy[256];
	__shared__ float Ixy[256];
	__shared__ float SumIxx[256];
	__shared__ float SumIyy[256];
	__shared__ float SumIxy[256];
	__shared__ float SumIxt[256];
	__shared__ float SumIyt[256];
	__shared__ float tempX;
	__shared__ float tempY;
    __shared__ float predictedX;
    __shared__ float predictedY;
	__shared__ float determinant;
	__shared__ float inverseDeterminant;
	__shared__ int breakCondition;
	
	int x   = threadIdx.x;
	int y   = threadIdx.y;
	int idx = blockIdx.x;
	int index = y*blockDim.x + x;
	
	if((index <169) && (status[blockIdx.x] != 0)){

			float previousX = ptsI[2*idx]  *pyramidScales;
			float previousY = ptsI[2*idx+1]*pyramidScales;
			
			if(index == 0){
				if(initialGuess) {
					tempX = 0.0f;
					tempY = 0.0f;
					predictedX = previousX;
					predictedY = previousY;
				}
				else {
					predictedX = ptsJ[2*idx];
					predictedY = ptsJ[2*idx+1];
					tempX = predictedX - previousX;
					tempY = predictedY - previousY;
				}
			}
			
			if((index > 0) &&(index < 88)){
				SumIxx[index + 168] = 0.0f;
				SumIyy[index + 168] = 0.0f;
				SumIxy[index + 168] = 0.0f;
				SumIxt[index + 168] = 0.0f;
				SumIyt[index + 168] = 0.0f;
			}
			
			
			
			Ix[index]  = (tex2D(texturePreviousFrame, previousX + x-5, previousY + y-6) - tex2D(texturePreviousFrame, previousX + x-7, previousY + y-6)) * 0.5f;
			Iy[index]  = (tex2D(texturePreviousFrame, previousX + x-6, previousY + y-5) - tex2D(texturePreviousFrame, previousX + x-6, previousY + y-7)) * 0.5f;
			Ixy[index] = tex2D(texturePreviousFrame, previousX + x-6, previousY + y-6);
			
			SumIxx[index] = Ix[index] * Ix[index];
			SumIyy[index] = Iy[index] * Iy[index];
			SumIxy[index] = Ix[index] * Iy[index];
			
			__syncthreads();
			
			
			if(index < 128){
				SumIxx[index] += SumIxx[index+128];
				SumIxy[index] += SumIxy[index+128];
				SumIyy[index] += SumIyy[index+128];
			}
			
			__syncthreads();
						
			if(index  < 64){
				SumIxx[index] += SumIxx[index+64];
				SumIxy[index] += SumIxy[index+64];
				SumIyy[index] += SumIyy[index+64];
			}
			
			__syncthreads();

			if(index < 32){
				SumIxx[index] += SumIxx[index+32];
				SumIxy[index] += SumIxy[index+32];
				SumIyy[index] += SumIyy[index+32];
			}
			
			__syncthreads();

			if(index < 16){
				SumIxx[index] += SumIxx[index+16];
				SumIxy[index] += SumIxy[index+16];
				SumIyy[index] += SumIyy[index+16];
			}
			
			__syncthreads();
			
			if(index < 8){
				SumIxx[index] += SumIxx[index+8];
				SumIxy[index] += SumIxy[index+8];
				SumIyy[index] += SumIyy[index+8];
			}
			
			__syncthreads();
			
			if(index < 4){
				SumIxx[index] += SumIxx[index+4];
				SumIxy[index] += SumIxy[index+4];
				SumIyy[index] += SumIyy[index+4];
			}
			
			__syncthreads();
			
			if(index < 2){
				SumIxx[index] += SumIxx[index+2];
				SumIxy[index] += SumIxy[index+2];
				SumIyy[index] += SumIyy[index+2];
			}
			
			__syncthreads();
			
			if(index == 0){
				SumIxx[index] += SumIxx[index+1];
				SumIxy[index] += SumIxy[index+1];
				SumIyy[index] += SumIyy[index+1];
				determinant = SumIxx[index] * SumIyy[index] - SumIxy[index] * SumIxy[index];
				inverseDeterminant = 1.0f/determinant;
				
				breakCondition = 0;
				if(determinant < 0.00001f) {
					status[idx] = 0;
					breakCondition = 1;
				}
			}
			
			__syncthreads();
			
			if(breakCondition == 1) {
				return;
			}
			
			
			for(int i = 0; i<10; i++){
				if(index == 0){
					if(predictedX < 0 || predictedX > width || predictedY < 0 || predictedY > scaledHeights) {
						status[idx] = 0;
						 breakCondition = 1;
					}
				}
				
				__syncthreads();

				if( breakCondition == 1) return;
				
				float It = tex2D(textureCurrentFrame, predictedX + x - 6 , predictedY + y - 6) - Ixy[index];
				SumIxt[index] = Ix[index] * It;
				SumIyt[index] = Iy[index] * It;
					
				__syncthreads();
				
				
				if(index < 128){
					SumIxt[index] += SumIxt[index+128];
					SumIyt[index] += SumIyt[index+128];
				}
				
				__syncthreads();
							
				if(index  < 64){
					SumIxt[index] += SumIxt[index+64];
					SumIyt[index] += SumIyt[index+64];
				}
				
				__syncthreads();

				if(index < 32){
					SumIxt[index] += SumIxt[index+32];
					SumIyt[index] += SumIyt[index+32];
				}

				__syncthreads();

				if(index < 16){
					SumIxt[index] += SumIxt[index+16];
					SumIyt[index] += SumIyt[index+16];
				}
				
				__syncthreads();
				
				if(index < 8){
					SumIxt[index] += SumIxt[index+8];
					SumIyt[index] += SumIyt[index+8];
				}
				
				__syncthreads();
				
				if(index < 4){
					SumIxt[index] += SumIxt[index+4];
					SumIyt[index] += SumIyt[index+4];
				}
				
				__syncthreads();
				
				if(index < 2){
					SumIxt[index] += SumIxt[index+2];
					SumIyt[index] += SumIyt[index+2];
				}
				
				__syncthreads();
				
				if(index == 0){
					SumIxt[index] += SumIxt[index+1];
					SumIyt[index] += SumIyt[index+1];
					float vx = inverseDeterminant * ( -SumIyy[index] * SumIxt[index] + SumIxy[index] * SumIyt[index]);
					float vy = inverseDeterminant * (  SumIxy[index] * SumIxt[index] - SumIxx[index] * SumIyt[index]);

					tempX += vx;
					tempY += vy;
					predictedX += vx;
					predictedY += vy;
					
					breakCondition = 0;
					if(fabsf(vx) < 0.01f && fabsf(vy) < 0.01f) breakCondition = 1;
				}
				__syncthreads();	
				
				if(breakCondition == 1) break;
				
			}
			
			if(index == 0){
				    if(level != 0) {
						predictedX += predictedX;
						predictedY += predictedY;

						tempX += tempX;
						tempY += tempY;
					}

				ptsJ[2*idx]	  = predictedX;
				ptsJ[2*idx+1] = predictedY;
			}
		}
	}



//---------7x7 Image Blur Kernal Implementation----------------------------

//Image Blur Row wise
__global__ void imageBlurRowKernel( unsigned char* imageout,int width,int pitch) 
{	

	const int Blocksize=248;
	int x = blockIdx.x*Blocksize+threadIdx.x -4;
	int pX = threadIdx.x;
	unsigned int y = blockIdx.y;
	
	__shared__ unsigned char imageBlock[Blocksize+8];	
	
	
	if((x >=0) || (x < width)) imageBlock[threadIdx.x]= tex2D(textureImage,x,y);
	
	else if((( x >= -4) && (x < 0)) || ((x >= width)) && (x < (width+4))) imageBlock[threadIdx.x]=0.0; 
	
	if( x >= (width-4)) return;

	__syncthreads();
	
	if( pX < 248 ){
		imageout[ y*pitch + x + 4]= 
		0.0076f*imageBlock[pX++]+
		0.0361f*imageBlock[pX++]+
		0.1096f*imageBlock[pX++]+
		0.2134f*imageBlock[pX++]+
		0.2666f*imageBlock[pX++]+
		0.2134f*imageBlock[pX++]+
		0.1096f*imageBlock[pX++]+
		0.0361f*imageBlock[pX++]+
		0.0076f*imageBlock[pX];	
		}
		
}

//Image Blur Column wise
__global__ void imageBlurColKernel( unsigned char* imageout,int width,int height,int pitch) 
{

    int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y =  blockIdx.y;
	
	if((x < width) && (BlurNoOfThreads*y < height)){
		for(int i=0; i<BlurNoOfThreads; i++){
			int yIndex = BlurNoOfThreads*y+i-4;
			float val =  0.0076f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.0361f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.1096f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.2134f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.2666f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.2134f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.1096f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.0361f*tex2D(textureRowBlurredImage,x,yIndex++);
				  val += 0.0076f*tex2D(textureRowBlurredImage,x,yIndex);
				imageout[ (BlurNoOfThreads*y+i) * pitch + x ] = val;									
		}
	}
		
}

/*--------------BB Overlap Kernel-----------------------------
Calculate fraction of overlap between the boxes of the grid and the bounding box 
*/
__global__ void bbOverLapKernal(float* bb,float* grid,int rowBBGrid,int colBBGrid,float* overlap)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx >= colBBGrid) return;

	float x11 = grid[rowBBGrid*idx   ];
	float y11 = grid[rowBBGrid*idx +1];
	float x12 = grid[rowBBGrid*idx +2];
	float y12 = grid[rowBBGrid*idx +3];

	float x21 = bb[0];
	float y21 = bb[1];
	float x22 = bb[2];
	float y22 = bb[3];

	float interSectWidth  = max((min(x12,x22) - max(x11,x21) + 1.0f),0.0f);
	float interSectHeight = max((min(y12,y22) - max(y11,y21) + 1.0f),0.0f);

	float intersection = interSectHeight * interSectWidth ; 
	float area1 = (x22 - x21 + 1.0f) * (y22 - y21 + 1.0f);
	float area2 = (x12 - x11 + 1.0f) * (y12 - y11 + 1.0f);

	overlap[idx] = intersection / (area1 + area2 - intersection);
   
}

/*Warp Image using a affine transform matrix*/
__global__ void transformKernel(unsigned char* warpedImage, int width, int height,int warpedImage_pitch, float* matrix, float bbPX,float bbPY, int bbH, int bbW) 
{	
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	if((x > bbW) || (y > bbH)) return;


	float x1 = bbPX + (float)x;
	float y1 = bbPY + (float)y;

	float xShift = matrix[2];
	float yShift = matrix[5];

	float u = x1 - xShift;
	float v = y1 - yShift;

	float x0 = (matrix[0]*u + matrix[1]*v + xShift);
	float y0 = (matrix[3]*u + matrix[4]*v + yShift);

	// read from texture and write to global memory
	warpedImage[ (int)(y1*warpedImage_pitch) + (int)x1] = tex2D(textureBlurredImage, x0, y0);// + randNoice[(int)y1*width + (int)x1];
}

//---------------Fern Implementation------------------------------------------------------

/*calculate feature patterns for positive samples*/
__global__ void calculatePatternsPositiveKernal(int* idxBoxs,int* patt,float* grid,float* featuresOffsets,int featuresOffsetStep,
										int noOfBoxes,int gridStep){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx >= noOfBoxes) return;

	float x = grid[idxBoxs[idx]*gridStep + 0];
	float y = grid[idxBoxs[idx]*gridStep + 1];
	float fx0,fx1,fy0,fy1;
	int scaleIndex  = grid[idxBoxs[idx]*gridStep + 4] * featuresOffsetStep;
	unsigned char f0,f1;
	int index;


	for(int tree=0; tree<10;tree++){
		index =0;
		int treeIndex = scaleIndex + 52*tree;
		for(int feature = 0;feature< 13;feature++){
			int featureIndex = treeIndex + 4*feature;
			fx0 = featuresOffsets[featureIndex    ];
			fy0 = featuresOffsets[featureIndex + 1];
			fx1 = featuresOffsets[featureIndex + 2];
			fy1 = featuresOffsets[featureIndex + 3];

			f0 = tex2D(textureWarpedImage,x+fx0,y+fy0); 
			f1 = tex2D(textureWarpedImage,x+fx1,y+fy1);

			index<<=1;
			if(f0>f1){
				index |= 1;
			}
		}
		patt[10*idx + tree] = index;
	}
}

/*Check the Varience of a given patch and if it is lower than a threshold status is set to zero*/
__global__ void calBoxVarienceKernal(int* idxBoxs,char* status,float* grid,int gridStep,float varienceThresh,int noOfBoxes){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx >= noOfBoxes) return;

	int gridIndex = idxBoxs[idx];

	float x1 = grid[gridIndex*gridStep + 0];
	float y1 = grid[gridIndex*gridStep + 1];
	float x2 = grid[gridIndex*gridStep + 2];
	float y2 = grid[gridIndex*gridStep + 3];
	float area = (y2-y1+1)*(x2-x1+1);

	
	float mX = (tex2D(textureIntregalImage,x2,y2)-tex2D(textureIntregalImage,x2,y1)
				-tex2D(textureIntregalImage,x1,y2)+tex2D(textureIntregalImage,x1,y1))/area;
	
	float mX2 = (tex2D(textureIntregalSQImage,x2,y2)-tex2D(textureIntregalSQImage,x2,y1)
				-tex2D(textureIntregalSQImage,x1,y2)+tex2D(textureIntregalSQImage,x1,y1))/area;

	if((mX2- mX*mX) >= varienceThresh)
	{
		status[idx] = 1;
	} 
}

/*Calculate Feature Patterens for Negative Samples*/
__global__ void calculatePatternsNegatveKernal(int* idxBoxs,char* status,int* patt,float* grid,float* featuresOffsets,int featuresOffsetStep,
										int noOfBoxes,int gridStep){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx >= noOfBoxes) return;

	if(status[idx] == 0) {
		for(int tree=0; tree<10;tree++){
			patt[10*idx + tree] = 0; }
		return;}

	float x = grid[idxBoxs[idx]*gridStep + 0];
	float y = grid[idxBoxs[idx]*gridStep + 1];
	float fx0,fx1,fy0,fy1;
	int scaleIndex  = grid[idxBoxs[idx]*gridStep + 4] * featuresOffsetStep;
	unsigned char f0,f1;
	int index;


	for(int tree=0; tree<10;tree++){
		index =0;
		int treeIndex = scaleIndex + 52*tree;
		for(int feature = 0;feature< 13;feature++){
			int featureIndex = treeIndex + 4*feature;		
			fx0 = featuresOffsets[featureIndex    ];
			fy0 = featuresOffsets[featureIndex + 1];
			fx1 = featuresOffsets[featureIndex + 2];
			fy1 = featuresOffsets[featureIndex + 3];

			f0 = tex2D(textureWarpedImage,x+fx0,y+fy0);
			f1 = tex2D(textureWarpedImage,x+fx1,y+fy1);

			index<<=1;
			if(f0>f1){
				index |= 1;
			}
		}
		patt[10*idx + tree] = index;
	}
}

/*Update weights using Negative samples*/
__global__ void updateNegativeKernal(float* weights,int* nP,int* nN,int* nX,int pattStep,
											 int noOfPats,float updateThreshNegative){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= noOfPats) return;
	
	int pattern[10];
	float confidenceVal =0.0f; 

	for(int tree =0; tree<pattStep; tree++){
		pattern[tree] = nX[idx*pattStep + tree];
		confidenceVal += weights[pattern[tree]*pattStep + tree];
	}
	if(confidenceVal >= updateThreshNegative){
		for(int tree =0; tree<pattStep; tree++){		
				nN[pattern[tree]*pattStep + tree]++;
				weights[pattern[tree]*pattStep + tree] = 
				((float)nP[pattern[tree]*pattStep + tree]/(float)(nP[pattern[tree]*pattStep + tree] + nN[pattern[tree]*pattStep + tree]));
		}
	}
}

/*Update weights using Positive samples*/
__global__ void updatePositiveKernal(float* weights,int* nP,int* nN,int* pX,int pattStep,
											 int noOfPats,float updateThreshPositive){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= noOfPats) return;
	
	int pattern[10];
	float confidenceVal =0.0f; 

	for(int tree =0; tree<pattStep; tree++){
		pattern[tree] = pX[idx*pattStep + tree];
		confidenceVal += weights[pattern[tree]*pattStep + tree];
	}
	if(confidenceVal <= updateThreshPositive){
		for(int tree =0; tree<pattStep; tree++){
			nP[pattern[tree]*pattStep + tree]++;
			weights[pattern[tree]*pattStep + tree] = 
				((float)nP[pattern[tree]*pattStep + tree]/(float)(nP[pattern[tree]*pattStep + tree] + nN[pattern[tree]*pattStep + tree]));
		}
	}
}

__global__ void getConfidences_fern3_Kernal(float* confidence,float* weights,int* nX,int pattStep,int pattColSize)
{
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= pattColSize) return;
	
	int pattern;
	float confidenceVal = 0.0f; 

	for(int tree =0; tree<pattStep; tree++){
		pattern = nX[idx*pattStep + tree];
		confidenceVal += weights[pattern*pattStep + tree];
	}

	confidence[idx] = confidenceVal;
}

/*Calculate the feature vectors for each bounding box in Grid*/
__global__ void calculatePatternsKernal(int* patt,float* grid,float* featuresOffsets,int featuresOffsetStep,
										int gridColSize,int gridStep){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx >= gridColSize) return; 

	float x = grid[idx*gridStep + 0];
	float y = grid[idx*gridStep + 1];
	float fx0,fx1,fy0,fy1;
	int scaleIndex  = (int)(grid[idx*gridStep + 4]) * featuresOffsetStep;
	unsigned char f0,f1;
	int index;


	for(int tree=0; tree<10;tree++){
		int treeIndex = scaleIndex + 52*tree;
		index =0;

		for(int feature = 0;feature< 13;feature++){
			int featureOndex = treeIndex + 4*feature;
			fx0 = featuresOffsets[featureOndex    ];
			fy0 = featuresOffsets[featureOndex + 1];
			fx1 = featuresOffsets[featureOndex + 2];
			fy1 = featuresOffsets[featureOndex + 3];

			f0 = tex2D(textureWarpedImage,x+fx0,y+fy0);//get pixel value of Feature Points 
			f1 = tex2D(textureWarpedImage,x+fx1,y+fy1);

			index<<=1;

			if(f0 > f1){
				index |= 1;
			}
		}
		patt[10*idx + tree] = index;
		/*This is a Integer value of 13 bit binary number xxxxxxxxxxxxx where x =0 if f0 > f1 and x =1 otherwise*/
	}

}

/*Calculate confidence value for each bounding box in the grid based on generated 13bit feature pattern*/
__global__ void getConfidencesFromTreeKernal(float* confidence,float* grid,float* weights,int* patt,int pattStep,
											 int gridColSize,int gridStep,float varienceThresh){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= gridColSize) return;
	
	int pattern;
	float x1,x2,y1,y2;
	float confidenceVal =0.0f;
	x1 = grid[idx*gridStep + 0];
	y1 = grid[idx*gridStep + 1];
	x2 = grid[idx*gridStep + 2];
	y2 = grid[idx*gridStep + 3];
	float area = (y2-y1+1)*(x2-x1+1);

	
	float mX = (tex2D(textureIntregalImage,x2,y2)-tex2D(textureIntregalImage,x2,y1)
				-tex2D(textureIntregalImage,x1,y2)+tex2D(textureIntregalImage,x1,y1))/area;
	
	float mX2 = (tex2D(textureIntregalSQImage,x2,y2)-tex2D(textureIntregalSQImage,x2,y1)
				-tex2D(textureIntregalSQImage,x1,y2)+tex2D(textureIntregalSQImage,x1,y1))/area;

	if((mX2- mX*mX) < varienceThresh) //Check whether the intensity of the box has significant varience(mX2- mX*mX) did to eliminate plain patches
	{
		confidence[idx] =0.0;
		return;
	}
	

	for(int tree =0; tree<10; tree++){
		pattern = patt[idx*pattStep + tree];
		confidenceVal += weights[pattern*10 + tree]; //confidence for the ten feature vectors are summed to get the confidence
	}

	confidence[idx] = confidenceVal;
}

//--------------- Sorting Result Kernel--------------------------------------------------//
__global__ void sort_fern(float* input,int* input_index,int size,int* sorted_index,int limit,float threshold)
{


    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx>=size)
	{
        return;
    }
   
    float val_2,val_1;
    val_1=input[idx];
   
    int tot=0;
	
	#pragma unroll
    for(int i=0;i<size;i++)
	{
        val_2=input[i];
        if(val_2>val_1 || (idx>i && val_2==val_1))
		{           
            tot++;
        }
        if(tot >= limit )
		{
            return;
        }
    }
    
	sorted_index[tot]=input_index[idx];
}

//-----------Normalised Cross Corelation Kernels ---------------------------------//

__global__ void ccorr_normed_nn(float *f1,float *f2,float *f3,int row_size,int f1_col,int f2_col)
{   
  int tid=threadIdx.y;
    __shared__ float f1_shared[1024];
    __shared__ float f2_shared[1024];   
    __shared__ float f1_f2_shared[1024];   

    if(tid<1012){
        int idx_2=floorf(blockIdx.y/f1_col)*row_size+2*tid;
        int idx_1=(blockIdx.y-floorf(blockIdx.y/f1_col)*f1_col)*row_size+2*tid;


        float a1=f1[idx_1];
        float b1=f2[idx_2];
        float a2=f1[idx_1+1];
        float b2=f2[idx_2+1];
   
        f1_f2_shared[tid]=a1*b1 +a2*b2;   
        f1_shared[tid]=a1*a1 +a2*a2 ;
        f2_shared[tid]=b1*b1 + b2*b2;
       
    }
    else if(tid==1012){
        int idx_2=floorf(blockIdx.y/f1_col)*row_size+2*tid;
        int idx_1=(blockIdx.y-floorf(blockIdx.y/f1_col)*f1_col)*row_size+2*tid;


        float a1=f1[idx_1];
        float b1=f2[idx_2];

        f1_f2_shared[tid]=a1*b1 ;   
        f1_shared[tid]=a1*a1 ;
        f2_shared[tid]=b1*b1 ;
    }

    else{

        f1_f2_shared[tid]=0.0;
        f2_shared[tid]=0.0;
        f1_shared[tid]=0.0;
    }

            __syncthreads();

			if(tid<512){
				f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+512];
				f1_shared[tid]=f1_shared[tid]+f1_shared[tid+512];
				f2_shared[tid]=f2_shared[tid]+f2_shared[tid+512];
			}

			__syncthreads();
			if(tid<256){
				f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+256];
				f1_shared[tid]=f1_shared[tid]+f1_shared[tid+256];
				f2_shared[tid]=f2_shared[tid]+f2_shared[tid+256];
			}

			__syncthreads();

			if(tid<128){
				f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+128];
				f1_shared[tid]=f1_shared[tid]+f1_shared[tid+128];
				f2_shared[tid]=f2_shared[tid]+f2_shared[tid+128];
			}

			__syncthreads();

            if(tid<64){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+64];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+64];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+64];
            }

            __syncthreads();

           
            if(tid<32){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+32];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+32];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+32];
            }
            __syncthreads();
           
            if(tid<16){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+16];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+16];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+16];
            }
            __syncthreads();
            if(tid<8){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+8];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+8];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+8];
            }
            __syncthreads();
            if(tid<4){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+4];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+4];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+4];
            }
            __syncthreads();
            if(tid<2){
                f1_f2_shared[tid]=f1_f2_shared[tid]+f1_f2_shared[tid+2];
                f1_shared[tid]=f1_shared[tid]+f1_shared[tid+2];
                f2_shared[tid]=f2_shared[tid]+f2_shared[tid+2];
            }
            __syncthreads();
           
            if(tid==0 ){
                f3[blockIdx.y]=(((f1_f2_shared[0]+f1_f2_shared[1])/sqrt((f1_shared[0]+f1_shared[1])*(f2_shared[0]+f2_shared[1])))+1)*0.5;
            }
   
}

//-----------Confidance checking Kernel ---------------------------------//
__global__ void calc_ConfIsin(int n,float* maxP,int* iP,float* maxPP,float* maxN,int* isin,float* conf1,float* conf2){
	
int idx = blockDim.y*blockIdx.y + threadIdx.y;
    if(idx>=n){
        return;
    }

    if(maxP[idx]>0.95){
            isin[3*idx]=1;
    }
    else{
        isin[3*idx]=-1;
    }
    isin[3*idx+1]=iP[idx];

    if(maxN[idx]>0.95){
        isin[3*idx+2]=1;
    }
    else{
        isin[3*idx+2]=-1;
    }

    // measure Relative Similarity
        float dN = 1 - maxN[idx];
        float dP = 1 - maxP[idx];
        conf1[idx] = dN / (dN + dP);

     // measure Conservative Similarity        ???
        dP = 1 - maxPP[idx];
        conf2[idx] = dN / (dN + dP);

}




//-----------Finding maximum of negative data Kernel ---------------------------------//
__global__ void get_maxN(float* input,int size,float* maxN){


   int idx = blockDim.y*blockIdx.y + threadIdx.y;
    if(idx>=size){
        return;
    }
    float val_1,val_2;
    val_1=input[idx];

    for(int i=0;i<size;i++){
        val_2=input[i];
       
        if(!(val_1>val_2 || (val_2==val_1 && i>=idx))){
            return;
        }       
    }

   
    maxN[0]=val_1;
}




//-----------Finding maximum of the total and first half of positive data Kernel ---------------------------------//
__global__ void get_maxP_maxPP(float* input,int size,float* maxP,float* maxPP,int* iP){

    int idx = blockDim.y*blockIdx.y + threadIdx.y;
    if(idx>=size){
        return;
    }
    float val_1,val_2,val_3;
    val_1=input[idx];
    val_3=input[idx];
    bool set1=1,set2=1;

    for(int i=0;i<size;i++){
        val_2=input[i];
        if(i<ceilf(size*0.5) ){
            if(!((val_3>val_2 || (val_2==val_3 && i>=idx)))){
                set1=0;
            }
        }
        if(!(val_1>val_2 || (val_2==val_1 && i>=idx))){
            set2=0;
        }
        if(!set1 && !set2){
            return;
        }
       

    }

    if(set1==1 && idx<ceilf(size*0.5)){
        maxPP[0]=val_3;
    }
    if(set2==1){
    maxP[0]=val_1;
    iP[0]=idx;
    }
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

CudaFunctions::CudaFunctions()
{

}

void CudaFunctions::freeMemory()
{
    for(int i=0; i < PYRAMID_LEVELS; i++) {
        hipFree(gpuImagePyramidPrevious[i]);
        hipFree(gpuImagePyramidCurrent[i]);
    }

    hipFree(gpuSmoothedXPrevious);
    hipFree(gpuSmoothedXCurrent);
    hipFree(gpuSmoothedPrevious);
    hipFree(gpuSmoothedCurrent);

    hipFreeArray(gpuArrayPyramidPrevious);
    hipFreeArray(gpuArrayPyramidCurrent);

    hipFree(gpuPreviousRGBImage);
    hipFree(gpuCurrentRGBImage);
    hipFree(gpu_status);


    delete [] status;
}

void CudaFunctions:: cudaGetPattern(int num,float* pattern,float *bb){

	int threadX = steps;
	int threadY = steps;

	int blockX = num;
	int blockY = 1;
	dim3 blocks(blockX,blockY);
	dim3 threads(threadX,threadY);
	
	float* gpu_patchout;
	float* gpu_bb;

	hipMalloc((void**)&gpu_patchout,patchSize*patchSize*num*sizeof(float));
	hipMalloc((void**)&gpu_bb,4*num*sizeof(float));
	cutilCheckMsg("Error in hipMalloc ...");
	
	hipMemcpy(gpu_bb,bb,4*num*sizeof(float),hipMemcpyHostToDevice);
	cutilCheckMsg("Error in cudaMemcpyToDevice...");	

	hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
	cutilCheckMsg("Error in warpedImage binding...");
	getPatternKernel<<<blocks,threads>>>(gpu_patchout,gpu_bb);
	cutilCheckMsg("Error in getPatternKernel...");
	hipMemcpy(pattern,gpu_patchout,patchSize*patchSize*num*sizeof(float),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in cudaMemcpyToHost...");

	hipFree(gpu_patchout);
	hipFree(gpu_bb);

}

/*Generate Intregal Images*/
void CudaFunctions::doIntregal(){
	
	hipBindTexture2D(0,&textureImage, gpu_image,&textureImage.channelDesc,width,height,gpu_image_pitch);
	cutilCheckMsg("Error in image binding...");

	rowIntegralKernel<<<h,512>>>(gpu_rowImage,w,h);
	cutilCheckMsg("Error in rowIntegralKernel...");

	rowIntegralSQKernel<<<h,512>>>(gpu_rowSQImage,w,h);
	cutilCheckMsg("Error in rowIntegralSQKernel...");
	
	colIntegralKernel<<<w,256>>>(gpu_rowImage,gpu_IntregalImage,w,h,gpu_IntregalImage_pitch/sizeof(float));
	cutilCheckMsg("Error in colIntegralKernel...");

	colIntegralSQKernel<<<w,256>>>(gpu_rowSQImage,gpu_IntregalSQImage,w,h,gpu_IntregalSQImage_pitch/sizeof(float));
	cutilCheckMsg("Error in colIntegralSQKernel...");

}


void CudaFunctions::freePointMemory(int nBoxes)
{
}


void CudaFunctions::initMemTrack(int _w,int _h)
{
	w = _w;
    h = _h;

    hipMalloc((void**)&gpuPreviousRGBImage, sizeof(char)*w*h*3);
    hipMalloc((void**)&gpuCurrentRGBImage, sizeof(char)*w*h*3);
    hipMalloc((void**)&gpuImagePyramidPrevious[0], sizeof(float)*w*h);
    hipMalloc((void**)&gpuImagePyramidCurrent[0], sizeof(float)*w*h);

    hipMalloc((void**)&gpuSmoothedXPrevious, sizeof(float)*w*h);
    hipMalloc((void**)&gpuSmoothedXCurrent, sizeof(float)*w*h);
    hipMalloc((void**)&gpuSmoothedPrevious, sizeof(float)*w*h);
    hipMalloc((void**)&gpuSmoothedCurrent, sizeof(float)*w*h);
	
	hipMallocArray(&gpuArrayPyramidPrevious, &texturePreviousFrame.channelDesc, w, h);
    hipMallocArray(&gpuArrayPyramidCurrent, &textureCurrentFrame.channelDesc, w, h);
	hipMallocArray(&gpuArrayPyramid, &textureCurrentFrame.channelDesc, w, h);

    texturePreviousFrame.normalized = 0;
    texturePreviousFrame.filterMode = hipFilterModeLinear;
    texturePreviousFrame.addressMode[0] = hipAddressModeClamp;
    texturePreviousFrame.addressMode[1] = hipAddressModeClamp;

    textureCurrentFrame.normalized = 0;
    textureCurrentFrame.filterMode = hipFilterModeLinear;
    textureCurrentFrame.addressMode[0] = hipAddressModeClamp;
    textureCurrentFrame.addressMode[1] = hipAddressModeClamp;

	texturePyramid.normalized = 0;
    texturePyramid.filterMode = hipFilterModeLinear;
    texturePyramid.addressMode[0] = hipAddressModeClamp;
    texturePyramid.addressMode[1] = hipAddressModeClamp;
	
	

    scaledWidths[0] = w;
    scaledHeights[0] = h;

    for(int i=1; i < PYRAMID_LEVELS; i++) {
        _w /= 2;
        _h /= 2;
        scaledWidths[i] = _w;
        scaledHeights[i] = _h;

        hipMalloc((void**)&gpuImagePyramidPrevious[i], sizeof(float)*_w*_h);
        hipMalloc((void**)&gpuImagePyramidCurrent[i], sizeof(float)*_w*_h);
    }
}

/*Process the initial Image frame for Tracking*/
void CudaFunctions::loadImageTrack(unsigned char *prev)
{
	
    int blocks1D = (w*h)/256 + (w*h % 256?1:0); // for greyscale
	
	hipMemcpy(gpuPreviousRGBImage, prev, w*h*3, hipMemcpyHostToDevice); 
	
	convertToGrey<<<blocks1D, 256>>>(gpuPreviousRGBImage, gpuImagePyramidPrevious[0], w*h);
	hipDeviceSynchronize();
    cutilCheckMsg("convertToGrey");
	
	for(int i=0; i < PYRAMID_LEVELS-1; i++) {
		
		//smoothX
		int length=252;	
		int blockSmoothXX = (scaledWidths[i]/length) + ((scaledWidths[i]%length)?1:0);
		int blockSmoothXY = scaledHeights[i];
		dim3 blocksSmoothX(blockSmoothXX,blockSmoothXY);

		hipMemcpy2DToArray(gpuArrayPyramid, 0, 0, gpuImagePyramidPrevious[i], 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice);
		hipBindTextureToArray(texturePyramid, gpuArrayPyramid);
        smoothX<<<blocksSmoothX, 256>>>(gpuSmoothedXPrevious, scaledWidths[i], scaledHeights[i]);
        hipDeviceSynchronize();
		cutilCheckMsg("smoothX here");
       
		//smoothY
		int blockSmoothYX = ( scaledWidths[i]/256) + (( scaledWidths[i]%256)?1:0);
		int blockSmoothYY = (scaledHeights[i]/SmoothYNoOfThreads) + ((scaledHeights[i]%SmoothYNoOfThreads)?1:0);
		dim3 blocksSmoothY(blockSmoothYX,blockSmoothYY);
		
		hipMemcpy2DToArray(gpuArrayPyramid, 0, 0, gpuSmoothedXPrevious, 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice);		
		hipBindTextureToArray(texturePyramid, gpuArrayPyramid);
		smoothY<<<blocksSmoothY, 256>>>(gpuSmoothedPrevious, scaledWidths[i], scaledHeights[i]);
        hipDeviceSynchronize();
		cutilCheckMsg("smoothY here");
		
		/////pyramid
		int nthreadsPyramid = 16;
		int blocksPyramidW = scaledWidths[i+1]/nthreadsPyramid + ((scaledWidths[i+1] % nthreadsPyramid)?1:0);
		int blocksPyramidH = scaledHeights[i+1]/nthreadsPyramid + ((scaledHeights[i+1] % nthreadsPyramid )?1:0);
		dim3 blocksPyramid(blocksPyramidW, blocksPyramidH);
		dim3 threadsPyramid(nthreadsPyramid, nthreadsPyramid);
		
		hipMemcpy2DToArray(gpuArrayPyramid, 0, 0, gpuSmoothedPrevious, 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice);	
        hipBindTextureToArray(texturePyramid, gpuArrayPyramid);
		pyrDownsample<<<blocksPyramid, threadsPyramid>>>(scaledWidths[i], scaledHeights[i], gpuImagePyramidPrevious[i+1], scaledWidths[i+1], scaledHeights[i+1]);
        hipDeviceSynchronize();
        cutilCheckMsg("pyrDownsample here");  
    }
}

void CudaFunctions::initPointSetMemoryTrack(int nBoxes)
{
	hipMalloc((void**)&gpu_ptsFB, sizeof(float)*nBoxes*200);
	hipMalloc((void**)&gpu_ptsJ, sizeof(float)*nBoxes*200);
	hipMalloc((void**)&gpu_ptsI, sizeof(float)*nBoxes*200);
    hipMalloc((void**)&gpu_status, sizeof(char)*nBoxes*100);
}

void CudaFunctions::run(intArrayStruct *confIdx,int2DArrayStruct patt,floatArrayStruct conf,float thresh,float var,float* ptsI,int nBoxes)
{	

    /*2 CUDA streams are created to carry on optical flow calculation and FERN detection functions simultaniously*/
	hipStreamCreate(&tracking_stream);
	hipStreamCreate(&fern_stream);

	int* gpu_patt;
	float* gpu_conf;
	hipMalloc((void**)&gpu_patt,patt.cols*patt.rows*sizeof(int));
	hipMalloc((void**)&gpu_conf,conf.size*sizeof(float));

	hipMemsetAsync(gpu_conf,0,conf.size*sizeof(float),fern_stream);

	//page locking data in main memory for hipMemcpyAsync 
	VirtualLock(patt.ptr, patt.cols*patt.rows*sizeof(int));
	VirtualLock(conf.ptr, conf.size*sizeof(float));
	VirtualLock(ptsI, 200*nBoxes*sizeof(float));
	VirtualLock(ptsJ,200*nBoxes*sizeof(float));
	VirtualLock(status,100*nBoxes*sizeof(char));

	hipMemcpyAsync(gpu_ptsI, ptsI, 200*nBoxes*sizeof(float), hipMemcpyHostToDevice,tracking_stream); 

	//tracking
	dim3 threadsTracking(13,13);
	int blocksTracking = 100;
   
////////////////////////////////fernDetection Detection/////////////////////////////////////////////////////////////////////////
	int threadsPatternsKernal = 256; 
	int blocksPatternsKernal = (gpu_grid.cols)/threadsPatternsKernal + (gpu_grid.cols % threadsPatternsKernal?1:0);
	int threadsConfidenceKernal = 256;
	int blocksConfidenceKernal = (gpu_grid.cols)/threadsConfidenceKernal + (gpu_grid.cols % threadsConfidenceKernal?1:0);

	hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
	cutilCheckMsg("Error in warpedImage binding..."); 
 	calculatePatternsKernal<<<blocksPatternsKernal,threadsPatternsKernal,0,fern_stream>>>(gpu_patt, gpu_grid.ptr, gpu_featuresOffsets.ptr,gpu_featuresOffsets.cols, gpu_grid.cols, gpu_grid.rows);
	cutilCheckMsg("Error in calculatePatternsKernal"); 

    hipStreamSynchronize(fern_stream);
	
	hipBindTexture2D(0,&textureIntregalImage,gpu_IntregalImage,&textureIntregalImage.channelDesc,width,height,gpu_IntregalImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalImage...");
	hipBindTexture2D(0,&textureIntregalSQImage,gpu_IntregalSQImage,&textureIntregalSQImage.channelDesc,width,height,gpu_IntregalSQImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalSQImage...");

	getConfidencesFromTreeKernal<<<blocksConfidenceKernal,threadsConfidenceKernal,0,fern_stream>>>(gpu_conf,gpu_grid.ptr,weights,gpu_patt,patt.rows,gpu_grid.cols,gpu_grid.rows,var);
	cutilCheckMsg("Error in getConfidencesFromTreeKernal(in Run)...");	
	
	hipMemcpyAsync(conf.ptr,gpu_conf, conf.size*sizeof(float),hipMemcpyDeviceToHost,fern_stream);
	hipMemcpyAsync(patt.ptr,gpu_patt, patt.cols*patt.rows*sizeof(int),hipMemcpyDeviceToHost,fern_stream);
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
 
    for(int i=0; i < PYRAMID_LEVELS-1; i++) {
		
		//Image is blured before optical flow calculation and the bluring matrix is decomposed to 2 vectors and smoothing is done in X and Y directions respectively 
		
		//X direction Smoothing
		int length=252;	
		int blockSmoothXX = (scaledWidths[i]/length) + ((scaledWidths[i]%length)?1:0);
		int blockSmoothXY = scaledHeights[i];
		dim3 blocksSmoothX(blockSmoothXX,blockSmoothXY);

		hipMemcpy2DToArrayAsync(gpuArrayPyramid, 0, 0, gpuImagePyramidCurrent[i], 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice,tracking_stream);	
		hipBindTextureToArray(texturePyramid, gpuArrayPyramid);
		smoothX<<<blocksSmoothX, 256,0,tracking_stream>>>(gpuSmoothedXCurrent, scaledWidths[i], scaledHeights[i]);
		hipStreamSynchronize(tracking_stream);
		cutilCheckMsg("smoothX ERROR");
       
		//Y direction Smoothing
		int blockSmoothYX = ( scaledWidths[i]/256) + (( scaledWidths[i]%256)?1:0);
		int blockSmoothYY = (scaledHeights[i]/SmoothYNoOfThreads) + ((scaledHeights[i]%SmoothYNoOfThreads)?1:0);
		dim3 blocksSmoothY(blockSmoothYX,blockSmoothYY);
		
		hipMemcpy2DToArrayAsync(gpuArrayPyramid, 0, 0, gpuSmoothedXCurrent, 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice,tracking_stream);	
		hipBindTextureToArray(texturePyramid, gpuArrayPyramid);
		smoothY<<<blocksSmoothY, 256,0,tracking_stream>>>( gpuSmoothedCurrent, scaledWidths[i], scaledHeights[i]);
		hipStreamSynchronize(tracking_stream);
		cutilCheckMsg("smoothY ERROR");
		
		
		//Create Image Pyramids
		int nthreadsPyramid = 16;
		int blocksPyramidW = scaledWidths[i+1]/nthreadsPyramid + ((scaledWidths[i+1] % nthreadsPyramid)?1:0);
		int blocksPyramidH = scaledHeights[i+1]/nthreadsPyramid + ((scaledHeights[i+1] % nthreadsPyramid )?1:0);
		dim3 blocksPyramid(blocksPyramidW, blocksPyramidH);
		dim3 threadsPyramid(nthreadsPyramid, nthreadsPyramid);

		hipMemcpy2DToArrayAsync(gpuArrayPyramid, 0, 0, gpuSmoothedCurrent, 
                           sizeof(float)*scaledWidths[i], sizeof(float)*scaledWidths[i], scaledHeights[i], hipMemcpyDeviceToDevice,tracking_stream);	
		hipBindTextureToArray(texturePyramid, gpuArrayPyramid);


		pyrDownsample<<<blocksPyramid, threadsPyramid,0,tracking_stream>>>(scaledWidths[i], scaledHeights[i], gpuImagePyramidCurrent[i+1],  scaledWidths[i+1], scaledHeights[i+1]);
		hipStreamSynchronize(tracking_stream);
        cutilCheckMsg("pyrDownsample ERROR");  
    }
	
	
    //set initial status of all points to Valid
    hipMemsetAsync(gpu_status, 1, sizeof(char)*100*nBoxes,tracking_stream);
	float *tempSwap;



    // Do the Optical Flow Calculation For the Image Pyramids from coarse to fine 	
    for(int l=PYRAMID_LEVELS-1; l >= 0; l--) {
        hipMemcpy2DToArrayAsync(gpuArrayPyramidPrevious, 0, 0, gpuImagePyramidPrevious[l], 
                           sizeof(float)*scaledWidths[l], sizeof(float)*scaledWidths[l], scaledHeights[l], hipMemcpyDeviceToDevice,tracking_stream);
        hipMemcpy2DToArrayAsync(gpuArrayPyramidCurrent, 0, 0, gpuImagePyramidCurrent[l], 
                            sizeof(float)*scaledWidths[l], sizeof(float)*scaledWidths[l], scaledHeights[l], hipMemcpyDeviceToDevice,tracking_stream);

		hipBindTextureToArray(texturePreviousFrame, gpuArrayPyramidPrevious);
		hipBindTextureToArray(textureCurrentFrame, gpuArrayPyramidCurrent);
        track<<<blocksTracking, threadsTracking,0,tracking_stream>>>(nBoxes, scaledWidths[l], scaledWidths[l], pyramidScales[l], l, (l == PYRAMID_LEVELS-1), gpu_ptsJ, gpu_status,gpu_ptsI);
		cutilCheckMsg("ERROR in RUN(Optical Flow Calculation)");  
		hipStreamSynchronize(tracking_stream);

		//swap Images: New Image are copied to gpuImagePyramidCurrent and after optical flow calculation 
		//it's pointers are swapped to make it gpuImagePyramidPrevious 
		tempSwap = gpuImagePyramidPrevious[l];
		gpuImagePyramidPrevious[l] = gpuImagePyramidCurrent[l];
		gpuImagePyramidCurrent[l] = tempSwap;
    }

	//copying predicted point set and status of validity to main memory
	hipMemcpyAsync(ptsJ, gpu_ptsJ, sizeof(float)*200*nBoxes, hipMemcpyDeviceToHost,tracking_stream);  
    hipMemcpyAsync(status, gpu_status, sizeof(char)*100*nBoxes, hipMemcpyDeviceToHost,tracking_stream);

	hipStreamSynchronize(tracking_stream);
	hipStreamDestroy(tracking_stream);

	hipStreamSynchronize(fern_stream);
	hipStreamDestroy(fern_stream);

	sort_data(conf.ptr,confIdx,conf.size,thresh);
	
	//unlocking locked memory
	VirtualUnlock(ptsI, 200*nBoxes*sizeof(float));
	VirtualUnlock(ptsJ,200*nBoxes*sizeof(float));
	VirtualUnlock(status,100*nBoxes*sizeof(char));
	VirtualUnlock(patt.ptr, patt.cols*patt.rows*sizeof(int));
	VirtualUnlock(conf.ptr, conf.size*sizeof(float));

	hipFree(gpu_patt);
	hipFree(gpu_conf);
}

/*calculate Optical flow in reverse direction*/
void CudaFunctions::run_FB(int nBoxes)
{
	dim3 threadsTracking(13,13);
	int blocksTracking = 100;

    for(int l=PYRAMID_LEVELS-1; l >= 0; l--) {

        hipMemcpy2DToArray(gpuArrayPyramidPrevious, 0, 0, gpuImagePyramidPrevious[l], 
                           sizeof(float)*scaledWidths[l], sizeof(float)*scaledWidths[l], scaledHeights[l], hipMemcpyDeviceToDevice);
        hipMemcpy2DToArray(gpuArrayPyramidCurrent, 0, 0, gpuImagePyramidCurrent[l], 
                            sizeof(float)*scaledWidths[l], sizeof(float)*scaledWidths[l], scaledHeights[l], hipMemcpyDeviceToDevice);
        
		hipBindTextureToArray(texturePreviousFrame, gpuArrayPyramidPrevious);
		hipBindTextureToArray(textureCurrentFrame, gpuArrayPyramidCurrent);

		//calculate optical flow in reverse due to previous swapping now the gpuImagePyramidCurrent points to the previous frame 
		track<<<blocksTracking, threadsTracking>>>(nBoxes, scaledWidths[l], scaledWidths[l], pyramidScales[l], l, (l == PYRAMID_LEVELS-1), gpu_ptsFB, gpu_status,gpu_ptsJ);
		cutilCheckMsg("ERROR in RUN FB(Optical Flow Calculation)");  
        hipDeviceSynchronize();
    }

}

/*Calculate Euclidian distance between actual point set(gpu_ptsI) and point
set predicted by backward optical flow calculatio (gpu_ptsFB)*/
void CudaFunctions::euclidianDistance(int nPts)
{
	float *gpu_distance;
	hipMalloc((void**)&gpu_distance,nPts*sizeof(float));

	int nThread = 256;
	int nBlock = nPts/nThread + ((nPts % nThread)? 1:0);

	calEuDistance<<<nBlock,nThread>>>(nPts,gpu_ptsI,gpu_ptsFB,gpu_distance);
	cutilCheckMsg("error in EUDISTANCE"); 

	hipMemcpy(fb,gpu_distance,nPts*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(gpu_distance);

}

/*Calculate Normalized Crosscorelation between actual point set(ptsI) 
and point set predicted by Optical Flow calculation (ptsJ)*/
void CudaFunctions::doNormalizedCrossCorr(int nPts, int Winsize)
{
	float *d_Ipts;
	float *d_Jpts;
	
	char* d_status;
	float* d_ncc;

	// allocate meory for points in device..	
	hipMalloc((void**)&d_Ipts,nPts*2*sizeof(float));
	hipMalloc((void**)&d_Jpts,nPts*2*sizeof(float));

	// copy points to device..
	hipMemcpy(d_Ipts,ptsI,nPts*2*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_Jpts,ptsJ,nPts*2*sizeof(float),hipMemcpyHostToDevice);

	// allocate memory for status in deivce..
	size_t statusSize = nPts*sizeof(char);
	hipMalloc((void**)&d_status,statusSize);

	// copy status to device..
	hipMemcpy(d_status,status,statusSize,hipMemcpyHostToDevice);


	// allocate memory for ncc in device..
	size_t nccSize = nPts*sizeof(float);
	hipMalloc((void**)&d_ncc,nccSize);

	// create threadblock..
	int nccThread = 64;
	int nccBlock = nPts/nccThread + ((nPts % nccThread)? 1:0);

	find_ncc_kernel<<<nccBlock,nccThread>>>(nPts,Winsize,d_Ipts,d_Jpts,d_status,d_ncc,w, h);
	cutilCheckMsg("NCC here");
	hipMemcpy(ncc,d_ncc,nccSize,hipMemcpyDeviceToHost);

	hipFree(d_Ipts);
	hipFree(d_Jpts);
	hipFree(d_status);
	hipFree(d_ncc);
}

/*Calculte Pointset(ptsI) for bounding Box*/
float* CudaFunctions::bb_points(float* bb,int numM,int numN,float margin)
{
	float tempX=bb[0]+margin;
	float tempY=bb[1]+margin;

	float stepW = (bb[2] - bb[0] - 2*margin) / (float)(numN - 1);
	float stepH = (bb[3] - bb[1] - 2*margin) / (float)(numM - 1);

	float* bbP = (float*)malloc(2*numN*numM*sizeof(float));
	float* tempbbP = bbP; 
	for(int i=0 ;i<numN;i++)
	{
		for(int j=0;j<numM;j++)
		{
			*tempbbP++ = tempX+i*stepW;
			*tempbbP++ = tempY+j*stepH;

		}
	}
	return bbP;

}

/*Calculate median of Given Dataset of size nPts*/
float CudaFunctions::median(float arr[],int nPts)
{
	if(nPts == 0)
		return 0;

	sortMedian(arr,nPts);
	int med=nPts/2;
	if(nPts%2==0)
		return (arr[med-1]+arr[med])/2;
	else
		return arr[med];

}

/*Calculate Pairwise Distance of given 2 datasets of size nPts*/
float CudaFunctions::pdist(float pts0[],float pts1[],int nPts)
{
	if(nPts == 0) return 1;

	int size = nPts*(nPts-1)/2;
	float *pdistArr= (float*)malloc(size*sizeof(float));
	int k=0;
	for(int i=0;i<nPts;i++){
		for(int j =i+1 ;j<nPts;j++){
			pdistArr[k]=sqrt(pow((pts1[2*i]-pts1[2*j]),2)+pow((pts1[2*i+1]-pts1[2*j+1]),2))/sqrt(pow((pts0[2*i]-pts0[2*j]),2)+pow((pts0[2*i+1]-pts0[2*j+1]),2));
			k++;
		}
	}
	float pdistMedian=median(pdistArr,k);
	free(pdistArr);
	return pdistMedian;	

}

/*Sort only to midpoint using bubble sort sufficent to get the mediant*/
void CudaFunctions::sortMedian(float arr[], int end)
{
	float t;
	for (int i=0;i<(end/2+1);i++){
		for(int j=i+1;j<end;j++){
			if(arr[i]>arr[j]){
				t=arr[i]; arr[i]=arr[j]; arr[j]=t;
			}
		}
	}

}

/*Predict the Next Bounding Box*/
void CudaFunctions::bb_predict(float* bb0,float* bb1, int nPts,float *medFB){
	
	float *pt0Crop = (float*)malloc(2*nPts*sizeof(float));
	float *pt1Crop = (float*)malloc(2*nPts*sizeof(float));
	float *temFb = (float*)malloc(nPts*sizeof(float));
	float *temNcc = (float*)malloc(nPts*sizeof(float));
	float *xArr = (float*)malloc(nPts*sizeof(float));
	float *yArr = (float*)malloc(nPts*sizeof(float));
   
	memcpy(temFb,fbIN,nPts*sizeof(float));
	memcpy(temNcc,nccIN,nPts*sizeof(float));

	float medianFbJ=median(temFb,nPts); //meidan of Foward Backward ERROR
    float medianNccJ=median(temNcc,nPts); //median of Normalized Crosscorelation
	
	*medFB = medianFbJ;
    
	//Choose Point wich have a less Foward Backward Error than medianFbJ and High Normalized Crosscorelation than medianNccJ
	int j=0;
	for (int i = 0; i < nPts; i++) {
		if(fbIN[i] <= medianFbJ && nccIN[i] >= medianNccJ){
			xArr[j] = ptsJIN[2*i]   - ptsIIN[2*i];
			yArr[j] = ptsJIN[2*i+1] - ptsIIN[2*i+1];
			pt0Crop[2*j] = ptsIIN[2*i];  pt0Crop[2*j+1] = ptsIIN[2*i+1];
			pt1Crop[2*j] = ptsJIN[2*i];  pt1Crop[2*j+1] = ptsJIN[2*i+1];
			j++;
		}
	}
	nPts = j;

	float medianDist = pdist(pt0Crop,pt1Crop,nPts); //pairwise didtance of pointset I and point set J (measure of scale change)
	float medianX = median(xArr,nPts); //median of X shift
    float medianY = median(yArr,nPts); //median of Y shift

    
	free(xArr);
	free(yArr);
	free(pt0Crop);
	free(pt1Crop);
	free(temFb);
	free(temNcc);
   
    float s1  = 0.5*(medianDist-1)*(float)(bb0[2] - bb0[0]); //scale change in X direction
    float s2  = 0.5*(medianDist-1)*(float)(bb0[3] - bb0[1]); //scale change in Y direction	
	
	/*Predicting Next Bounding Box 0.5 is added to compensate for the loss of precision in Float to Int conversion*/
	bb1[0]      = floorf(bb0[0] +medianX + 0.5 -s1);
	bb1[1]     	= floorf(bb0[1] +medianY + 0.5 -s2);
	bb1[2]  	= floorf(bb0[2] +medianX + 0.5 +s1);
	bb1[3]		= floorf(bb0[3] +medianY + 0.5 +s2);
}


void CudaFunctions::doLK(unsigned char *cur,float* bbIn,float* bbOut,float *medFB,intArrayStruct *confIdx, int2DArrayStruct patt,floatArrayStruct conf,float thresh,float var)
{
	/*100 (10 x 10) sample points from the bounding box is taken*/
	int numM =10; 
	int numN = 10;
	float margin =5.0;
	int nPts = 100;

	ptsI = bb_points(bbIn,numM,numN,margin); //get cordinates of sample points x(n) = ptsI[2n], y[n] = ptsI[2n+1]
	ptsJ = (float*) malloc(2*nPts*sizeof(float));

	status = (char*)  malloc(nPts*sizeof(char));
	ncc    = (float*) malloc(nPts*sizeof(float));
	fb     = (float*) calloc(nPts,sizeof(float));
	memset(status,1,nPts*sizeof(char));

	ptsIIN = (float*) malloc(2*nPts*sizeof(float));
	ptsJIN = (float*) malloc(2*nPts*sizeof(float));
	fbIN   = (float*) malloc(nPts*sizeof(float));
	nccIN  = (float*) malloc(nPts*sizeof(float));


	run(confIdx,patt,conf,thresh,var,ptsI); //calculate optical flow
	run_FB(); //Calculate optical flow backward
	doNormalizedCrossCorr(nPts); //calculate Normalize Crosscorelation	
	euclidianDistance(nPts); //Calculate Euclidian Distance 

	
	

	int j =0; 
	for(int i=0;i<100;i++)
	{
		if (status[i] == 1) 
		{

			ptsIIN[2*j]	= ptsI[2*i];	// x val
			ptsIIN[2*j+1]= ptsI[2*i+1];	// y val

			ptsJIN[2*j]		= ptsJ[2*i];	// x val
			ptsJIN[2*j+1]	= ptsJ[2*i+1];	// y val

			fbIN[j]  =  fb[i];
			nccIN[j++] = ncc[i];
		} 
	}
	nPts = j;

	free(ptsI);
	free(ptsJ);
	free(fb);
	free(ncc);
	free(status);
	

	bb_predict(bbIn, bbOut ,nPts,medFB); //predict the next bounding box


	free(ptsIIN);
	free(ptsJIN);
	free(fbIN);
	free(nccIN);	
}

///////////////////////////-----------CUDA FERN Definitions---------------------//////////////////////////////


void CudaFunctions::initializeFern(float2DArrayStruct grid, float2DArrayStruct featureOffsets, int w, int h)
{	
	initMem_fern(w,h);
	loadGrid(grid);
	loadFeatureOffsets(featureOffsets);
}

/*Initialize Fern's Memory*/
void CudaFunctions::initMem_fern(int w,int h)
{
	width  = w;
	height = h;
	
	hipMalloc((void**)&weights,81920*sizeof(float));
	cutilCheckMsg("Error in creating weights...");
	hipMalloc((void**)&nP,81920*sizeof(int));
	cutilCheckMsg("Error in creating nP...");
	hipMalloc((void**)&nN,81920*sizeof(int));
	cutilCheckMsg("Error in creating nN...");

	hipMemset(weights,0,81920*sizeof(float));
	cutilCheckMsg("Error in initializing weights...");
	hipMemset(nP,0,81920*sizeof(int));
	cutilCheckMsg("Error in initializing nP...");
	hipMemset(nN,0,81920*sizeof(int));
	cutilCheckMsg("Error in initializing nN...");


	hipMallocPitch((void**)&gpu_image,&gpu_image_pitch,width*sizeof(unsigned char),height);
	hipMallocPitch((void**)&gpu_blurImageRow,&gpu_blurImageRow_pitch,width*sizeof(unsigned char),height);
	hipMallocPitch((void**)&gpu_blurImage,&gpu_blurImage_pitch,width*sizeof(unsigned char),height);
	hipMallocPitch((void**)&gpu_warpedImage,&gpu_warpedImage_pitch,width*sizeof(unsigned char),height);
	hipMalloc((void**)&gpu_rowImage,width*height*sizeof(float));
	hipMallocPitch((void**)&gpu_IntregalImage,&gpu_IntregalImage_pitch,width*sizeof(float),height);
	hipMalloc((void**)&gpu_rowSQImage,width*height*sizeof(float));
	hipMallocPitch((void**)&gpu_IntregalSQImage,&gpu_IntregalSQImage_pitch,width*sizeof(float),height);
	cutilCheckMsg("Error in hipMalloc ...");

	textureImage.normalized = 0;
	textureImage.filterMode = hipFilterModePoint;
    textureImage.addressMode[0] = hipAddressModeClamp;
    textureImage.addressMode[1] = hipAddressModeClamp;
	
	textureBlurredImage.normalized = 0;
	textureBlurredImage.filterMode = hipFilterModePoint;
    textureBlurredImage.addressMode[0] = hipAddressModeBorder;
    textureBlurredImage.addressMode[1] = hipAddressModeBorder;

	textureWarpedImage.normalized = 0;
	textureWarpedImage.filterMode = hipFilterModePoint;
    textureWarpedImage.addressMode[0] = hipAddressModeClamp;
    textureWarpedImage.addressMode[1] = hipAddressModeClamp;

	textureRowBlurredImage.normalized = 0;
	textureRowBlurredImage.filterMode = hipFilterModePoint;
	textureRowBlurredImage.addressMode[0] = hipAddressModeClamp;
	textureRowBlurredImage.addressMode[1] = hipAddressModeClamp;

	textureIntregalImage.normalized = 0;
	textureIntregalImage.filterMode = hipFilterModePoint;
    textureIntregalImage.addressMode[0] = hipAddressModeClamp;
    textureIntregalImage.addressMode[1] = hipAddressModeClamp;

	textureIntregalSQImage.normalized = 0;
	textureIntregalSQImage.filterMode = hipFilterModePoint;
    textureIntregalSQImage.addressMode[0] = hipAddressModeClamp;
    textureIntregalSQImage.addressMode[1] = hipAddressModeClamp;

}

/*Load Grid Points from Main Memory to GPU Global memory*/
void CudaFunctions::loadGrid(float2DArrayStruct grid)
{

	gpu_grid.cols = grid.cols;
	gpu_grid.rows = grid.rows;

	hipMalloc((void**)&gpu_grid.ptr,grid.cols*grid.rows*sizeof(float));
	hipMemcpy(gpu_grid.ptr, grid.ptr, grid.cols*grid.rows*sizeof(float), hipMemcpyHostToDevice);	
	cutilCheckMsg("Error in loading Grid...");
	
}

/*Copy Generated Feature Offsets to GPU Memory*/
void CudaFunctions::loadFeatureOffsets(float2DArrayStruct featureOffsets)
{
	gpu_featuresOffsets.cols = featureOffsets.cols; //520 = 10 Feature Vectors x 13 Features per Vector x 4 Cordinates(2 points) per Feature
	gpu_featuresOffsets.rows = featureOffsets.rows; //num of bounding box sizes

	hipMalloc((void**)&gpu_featuresOffsets.ptr,featureOffsets.cols*featureOffsets.rows*sizeof(float));
	hipMemcpy(gpu_featuresOffsets.ptr, featureOffsets.ptr, featureOffsets.cols*featureOffsets.rows*sizeof(float), hipMemcpyHostToDevice);
	cutilCheckMsg("Error in loading featureOffsets...");

}

/*Load Image initialy*/
void CudaFunctions::loadImage(unsigned char* imageData)
{
	int blocks1D = (width*h)/256 + (width*height % 256?1:0); //threads for greyscale

	
	hipMemcpy(gpuCurrentRGBImage, imageData, width*height*3, hipMemcpyHostToDevice); //copy RGB image to GPU main memory

	convertToGrey<<<blocks1D, 256>>>(gpuCurrentRGBImage, gpuImagePyramidCurrent[0], width*height);
	hipDeviceSynchronize();
    cutilCheckMsg("convertToGrey");

	dim3 threadFloatToCharKernal(16,16);
	int blockY = (height/16) + ((height%16)?1:0);
	int blockX = (width/16) + ((width%16)?1:0);;
	dim3 blockFloatToCharKernal(blockX,blockY);
	floatToCharKernal<<<blockFloatToCharKernal, threadFloatToCharKernal>>>(gpuImagePyramidCurrent[0],gpu_image,height,width,gpu_image_pitch);
	cutilCheckMsg("convert FloatToChar");

	doImageBlurRow(width,  height);
	cutilCheckMsg("Error in gpu_blurImage binding...");

	doImageBlurCol(width, height);

	hipMemcpy2D(gpu_warpedImage,gpu_warpedImage_pitch,gpu_blurImage,gpu_blurImage_pitch,sizeof(unsigned char)*width,
				height,hipMemcpyDeviceToDevice);
	cutilCheckMsg("Error in warpedImage loading...");
	
	doIntregal();

}

void CudaFunctions:: doImageBlurRow( int w, int h)
{

	int length=248;
	int threadX = 256;
	int threadY = 1;
	
	int blockX = (w/length) + ((w%length)?1:0);
	int blockY = h;
	
	dim3 blocks(blockX,blockY);
	dim3 threads(threadX,threadY);	

	hipBindTexture2D(0,&textureImage, gpu_image,&textureImage.channelDesc,width,height,gpu_image_pitch);
	cutilCheckMsg("Error in image binding...");
	imageBlurRowKernel<<<blocks,threads>>>(gpu_blurImageRow,w,gpu_blurImageRow_pitch/sizeof(unsigned char));
	cutilCheckMsg("Error in imageBlurRowKernel...");

}

void CudaFunctions:: doImageBlurCol(int w, int h)
{


	int length=256;
	int blockX = (w/length) + ((w%length)?1:0);
	int blockY = (h/BlurNoOfThreads) + ((h%BlurNoOfThreads)?1:0);
	dim3 blocks(blockX,blockY);

	hipBindTexture2D(0,&textureRowBlurredImage, gpu_blurImageRow,&textureRowBlurredImage.channelDesc,width,height,gpu_blurImageRow_pitch);	
	imageBlurColKernel<<<blocks,length>>>(gpu_blurImage,w,h,gpu_blurImage_pitch/sizeof(unsigned char));
	cutilCheckMsg("Error in imageBlurColKernel...");

}


void CudaFunctions::calcBBOverLapWithGrid(float* bb,int sizeBB,float* overlap)
{

	float *gpu_bb;
	float *gpu_overlap;

	hipMalloc((void**)&gpu_bb, sizeof(float)*sizeBB*4);
	hipMemcpy(gpu_bb, bb, 4*sizeBB*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&gpu_overlap, sizeof(float)*gpu_grid.cols*sizeBB);

	
	int threadbbOverLapKernal = 256;
	int blockbbOverLapKernal = (gpu_grid.cols)/threadbbOverLapKernal + (gpu_grid.cols % threadbbOverLapKernal?1:0); // for greyscale

	bbOverLapKernal<<<blockbbOverLapKernal,threadbbOverLapKernal>>>(gpu_bb,gpu_grid.ptr,gpu_grid.rows,gpu_grid.cols,gpu_overlap);
	cutilCheckMsg("Error in BBOVERLAP Kernal...");
	
	hipMemcpy( overlap,gpu_overlap,gpu_grid.cols*sizeBB*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(gpu_bb);
	hipFree(gpu_overlap);		
   
}

void CudaFunctions::fern5Positive(int2DArrayStruct* patt,float* bb,float2DArrayStruct matrix,intArrayStruct idxBoxs)
{

	int* gpu_patt;
	int* gpu_idxBoxs;
	int numWarps = matrix.cols;
	float* gpu_matrix;
	int bbW,bbH;
	bbW = (int)(bb[2] - bb[0] + 1.0);
	bbH = (int)(bb[3] - bb[1] + 1.0);

	hipMalloc((void**)&gpu_matrix,matrix.cols*matrix.rows*sizeof(float));
	hipMalloc((void**)&gpu_patt,patt->cols*patt->rows*sizeof(int));
	hipMalloc((void**)&gpu_idxBoxs,idxBoxs.size*sizeof(int));
	cutilCheckMsg("Error in hipMalloc fern5Positive...");

	hipMemcpy(gpu_idxBoxs,idxBoxs.ptr,idxBoxs.size*sizeof(int),hipMemcpyHostToDevice); 	
	cutilCheckMsg("Error in memcpy fern5Positive idxBoxs...");

	hipMemcpy(gpu_matrix,matrix.ptr,matrix.cols*matrix.rows*sizeof(float),hipMemcpyHostToDevice);
	cutilCheckMsg("Error in memcpy fern5Positive matrix...");

	int threadWarpX = 16;
	int threadWarpY = 16;

	int blockWarpX = (bbW/threadWarpX) + ((bbW%threadWarpX)?1:0);
	int blockWarpY = (bbH/threadWarpY) + ((bbH%threadWarpY)?1:0);

	dim3 blocks(blockWarpX,blockWarpY);
	dim3 threads(threadWarpX,threadWarpY);

	int threadsPatternsKernal = 256;
	int blocksPatternsKernal = (idxBoxs.size)/threadsPatternsKernal + (idxBoxs.size % threadsPatternsKernal?1:0);

	hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
	cutilCheckMsg("Error in warpedImage binding...");
	calculatePatternsPositiveKernal<<<blocksPatternsKernal,threadsPatternsKernal>>>(gpu_idxBoxs, gpu_patt, gpu_grid.ptr, gpu_featuresOffsets.ptr,gpu_featuresOffsets.cols,idxBoxs.size, gpu_grid.rows);
	cutilCheckMsg("Error in calculatePatternsPositiveKernal...");

	for(int i=0;i<numWarps;i++)
	{

		hipBindTexture2D(0,&textureBlurredImage, gpu_blurImage,&textureBlurredImage.channelDesc,width,height,gpu_blurImage_pitch);
		cutilCheckMsg("Error in blurImage binding...");
		transformKernel<<<blocks,threads>>>(gpu_warpedImage,width,height,gpu_warpedImage_pitch/sizeof(unsigned char),(gpu_matrix+i*9),bb[0],bb[1],bbW,bbH);
		hipDeviceSynchronize();
		cutilCheckMsg("Error in WARP Kernal...");

		hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
		cutilCheckMsg("Error in warpedImage binding...");
		calculatePatternsPositiveKernal<<<blocksPatternsKernal,threadsPatternsKernal>>>(gpu_idxBoxs, (gpu_patt+(i+1)*10*idxBoxs.size), gpu_grid.ptr, gpu_featuresOffsets.ptr,gpu_featuresOffsets.cols,idxBoxs.size, gpu_grid.rows);
		cutilCheckMsg("Error in calculatePatternsPositiveKernal...");
	}



	hipMemcpy(patt->ptr,gpu_patt,patt->cols*patt->rows*sizeof(int),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memcpy fern5Positive patt...");
		
	hipFree(gpu_patt);
	hipFree(gpu_idxBoxs);
	hipFree(gpu_matrix);

}

void CudaFunctions::fern5Negative(int2DArrayStruct patt,char* status, int *idxBoxs, float varienceThresh)
{
	int* gpu_patt;
	int* gpu_idxBoxs;
	char* gpu_status;
	
	hipMalloc((void**)&gpu_patt,patt.cols*patt.rows*sizeof(int));
	cutilCheckMsg("Error in hipMalloc gpu_patt fern5Negative...");
	hipMalloc((void**)&gpu_idxBoxs,patt.cols*sizeof(int));
	cutilCheckMsg("Error in hipMalloc gpu_idxBoxs fern5Negative...");
	hipMalloc((void**)&gpu_status,patt.cols*sizeof(char));
	cutilCheckMsg("Error in hipMalloc gpu_status fern5Negative...");

	hipMemset(gpu_status,0,patt.cols*sizeof(char));
	cutilCheckMsg("Error in memset gpu_status fern5Negative...");

	hipMemcpy(gpu_idxBoxs,idxBoxs,patt.cols*sizeof(int),hipMemcpyHostToDevice); 
	cutilCheckMsg("Error in memcpy fern5Negative idxBoxs...");

	int threadsPatternsKernal = 256;
	int blocksPatternsKernal = (patt.cols)/threadsPatternsKernal + (patt.cols % threadsPatternsKernal?1:0);
	int threadsBoxVarienceKernal = 256;
	int blocksBoxVarienceKernal = (patt.cols)/threadsBoxVarienceKernal + (patt.cols % threadsBoxVarienceKernal?1:0);

	hipBindTexture2D(0,&textureIntregalImage,gpu_IntregalImage,&textureIntregalImage.channelDesc,width,height,gpu_IntregalImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalImage...");
	hipBindTexture2D(0,&textureIntregalSQImage,gpu_IntregalSQImage,&textureIntregalSQImage.channelDesc,width,height,gpu_IntregalSQImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalSQImage...");
	calBoxVarienceKernal<<<blocksBoxVarienceKernal,threadsBoxVarienceKernal>>>(gpu_idxBoxs,gpu_status,gpu_grid.ptr,gpu_grid.rows,varienceThresh, patt.cols);
	cutilCheckMsg("Error in calBoxVarienceKernal...");

	hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
	cutilCheckMsg("Error in warpedImage binding...");
	calculatePatternsNegatveKernal<<<blocksPatternsKernal,threadsPatternsKernal>>>(gpu_idxBoxs,gpu_status,gpu_patt, gpu_grid.ptr, gpu_featuresOffsets.ptr,gpu_featuresOffsets.cols, patt.cols, gpu_grid.rows);
	cutilCheckMsg("Error in calculatePatternsNegatveKernal...");
	
	hipMemcpy(patt.ptr,gpu_patt,patt.cols*patt.rows*sizeof(int),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memcpy fern5Positive patt...");
	hipMemcpy(status,gpu_status,patt.cols*sizeof(char),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memcpy fern5Positive status...");	
	

	hipFree(gpu_patt);
	hipFree(gpu_idxBoxs);
	hipFree(gpu_status);	
}

void CudaFunctions::fernLearning(int2DArrayStruct pX, int2DArrayStruct nX,float threshPositive,float threshNegative,int bootStrap)
{
	int threadsUpdate = 256;
	if(pX.cols > 0)
	{
		int *gpu_pX;
		int blocksUpdatePositive = (pX.cols)/threadsUpdate + (pX.cols % threadsUpdate?1:0);

		hipMalloc((void**)&gpu_pX,pX.cols*pX.rows*sizeof(int));

		hipMemcpy(gpu_pX, pX.ptr, pX.cols*pX.rows*sizeof(int),hipMemcpyHostToDevice);
		cutilCheckMsg("Error in memcpy pX fernLearning...");
	
		for(int i=0; i<bootStrap; i++)
		{
			updatePositiveKernal<<<blocksUpdatePositive,threadsUpdate>>>(weights,nP,nN,gpu_pX,pX.rows,pX.cols,threshPositive);
			hipDeviceSynchronize();
			cutilCheckMsg("Error in updatePositiveKernal...");
		}
		hipFree(gpu_pX);
	}


	if(nX.cols > 0)
	{
		int *gpu_nX;

		int blocksUpdateNegative = (nX.cols)/threadsUpdate + (nX.cols % threadsUpdate?1:0);

		hipMalloc((void**)&gpu_nX, nX.cols*nX.rows*sizeof(int));	

		hipMemcpy(gpu_nX,nX.ptr, nX.cols*nX.rows*sizeof(int),hipMemcpyHostToDevice);
		cutilCheckMsg("Error in memcpy nX fernLearning...");

		for(int i=0; i<bootStrap; i++)
		{
			updateNegativeKernal<<<blocksUpdateNegative,threadsUpdate>>>(weights,nP,nN,gpu_nX,nX.rows,nX.cols,threshNegative);
			hipDeviceSynchronize();
			cutilCheckMsg("Error in updateNegativeKernal...");
			
		}
		hipFree(gpu_nX);

	}
}

void CudaFunctions::fern3(int2DArrayStruct nX,float* confidences)
{
	int threadsfern3 = 256;
	int *gpu_nX;
	float* gpu_confidences;

	hipMalloc((void**)&gpu_confidences,nX.cols*sizeof(float));
	hipMalloc((void**)&gpu_nX,nX.cols*nX.rows*sizeof(int));

	int blocksfern3 = (nX.cols)/threadsfern3 + (nX.cols % threadsfern3?1:0);

	//std::cout<<nX.cols<<"\t"<<nX.rows<<"\n";
	hipMemcpy(gpu_nX,nX.ptr,nX.cols*nX.rows*sizeof(int),hipMemcpyHostToDevice);
	cutilCheckMsg("Error in memcpy nX fern3...");

	getConfidences_fern3_Kernal<<<blocksfern3,threadsfern3>>>(gpu_confidences,weights,gpu_nX,nX.rows,nX.cols);
	cutilCheckMsg("Error in getConfidences_fern3_Kernal...");

	hipMemcpy(confidences,gpu_confidences,nX.cols*sizeof(float),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memcpy confidences fern3...");
	
	hipFree(gpu_confidences);		

}

/*Evaluate all the boxes generated by the grid using FERN and find confidences for them to be the object*/
void CudaFunctions::fernDetection(int2DArrayStruct patt,floatArrayStruct conf,intArrayStruct* tconfidx,float pthresh,float varienceThresh)
{
	int* gpu_patt;
	float* gpu_conf;
	

	hipMalloc((void**)&gpu_patt,patt.cols*patt.rows*sizeof(int));
	hipMalloc((void**)&gpu_conf,conf.size*sizeof(float));
	hipMemset(gpu_conf,0,conf.size*sizeof(float));	

	int threadsPatternsKernal = 192;
	int blocksPatternsKernal = (gpu_grid.cols)/threadsPatternsKernal + (gpu_grid.cols % threadsPatternsKernal?1:0);
	int threadsConfidenceKernal = 256;
	int blocksConfidenceKernal = (gpu_grid.cols)/threadsConfidenceKernal + (gpu_grid.cols % threadsConfidenceKernal?1:0);


	hipBindTexture2D(0,&textureWarpedImage, gpu_warpedImage,&textureWarpedImage.channelDesc,width,height,gpu_warpedImage_pitch);
	cutilCheckMsg("Error in warpedImage binding...");
	calculatePatternsKernal<<<blocksPatternsKernal,threadsPatternsKernal>>>(gpu_patt, gpu_grid.ptr, gpu_featuresOffsets.ptr,gpu_featuresOffsets.cols, gpu_grid.cols, gpu_grid.rows);
	cutilCheckMsg("Error in calculatePatternsKernal...");
	hipDeviceSynchronize();

	hipBindTexture2D(0,&textureIntregalImage,gpu_IntregalImage,&textureIntregalImage.channelDesc,width,height,gpu_IntregalImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalImage...");
	hipBindTexture2D(0,&textureIntregalSQImage,gpu_IntregalSQImage,&textureIntregalSQImage.channelDesc,width,height,gpu_IntregalSQImage_pitch);
	cutilCheckMsg("Error in BindTexture from gpu_IntregalSQImage...");
	getConfidencesFromTreeKernal<<<blocksConfidenceKernal,threadsConfidenceKernal>>>(gpu_conf,gpu_grid.ptr,weights,gpu_patt,patt.rows,gpu_grid.cols,gpu_grid.rows,varienceThresh);
	cutilCheckMsg("Error in getConfidencesFromTree Kernal...");


	hipMemcpy(patt.ptr,gpu_patt, patt.cols*patt.rows*sizeof(int),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memecpy patt...");
	hipMemcpy(conf.ptr,gpu_conf, conf.size*sizeof(float),hipMemcpyDeviceToHost);
	cutilCheckMsg("Error in memecpy conf...");

	sort_data(conf.ptr,tconfidx,conf.size,pthresh);

	hipFree(gpu_patt);
	hipFree(gpu_conf);

}

void CudaFunctions::sort_data(float *data,intArrayStruct *out, int input_size, float threshold)
{

	float *cuda_in_array;
    int *cuda_in_index;
    int *cuda_out_index;

    float* data_final=(float*)malloc(sizeof(float)*input_size);
    int* index_final=(int*)malloc(sizeof(int)*input_size);

	int limit = out->size;
    int a=0;
    for(int i=0;i<input_size;i++)
	{
        if(data[i]>threshold)
		{
            data_final[a]=data[i];
            index_final[a]=i;

			a++;
        }
    }
   
	if(a<limit)
		limit=a;

	if(limit==0)
	{
		out->size = 0;
		return;
	}

    hipMalloc( (void**)&cuda_in_array, a*sizeof(float));
    hipMemcpy( cuda_in_array, data_final,a*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc( (void**)&cuda_in_index, a*sizeof(int));
    hipMemcpy( cuda_in_index, index_final,a*sizeof(int), hipMemcpyHostToDevice);


    hipMalloc( (void**)&cuda_out_index, (limit)*sizeof(int));
   
    int block_size =256;
    dim3 threads(block_size,1);
    dim3 grid((input_size+a-1) / threads.x,1/ threads.y);

    sort_fern<<< grid, threads>>>(cuda_in_array,cuda_in_index,a,cuda_out_index,limit,threshold);
    hipDeviceSynchronize();
	
	hipMemcpyAsync(out->ptr, cuda_out_index,limit* sizeof(int), hipMemcpyDeviceToHost);
	
	
	hipFree(cuda_out_index);
    hipFree(cuda_in_array);
    hipFree(cuda_in_index);
	
	free(data_final);
	free(index_final);
	out->size = limit;

	
}

//_________Nearest Neighbor Confidance Calculation___________
//Distance between given set of patches and existing �pex� and �nex� data stored in TLD structure are calculated to find out 
//which of the given patches have the closest resemblance to �pex� and �nex�.
void CudaFunctions::calc_tldNN(float* x,int n,int M,float* pex,int N1,float* nex,int N2,float* conf1,float* conf2,int* isin)
{
    for (int i=0;i<3*n;i++)
        isin[i]= -1;

     
    if(N1==0)
    {// IF positive examples in the model are not defined THEN everything is negative
        for (int i=0;i<n;i++)
        {
            conf1[i]=0;
            conf2[i]=0; 
        }
        return;
    }

    if(N2==0)
    {// IF negative examples in the model are not defined THEN everything is positive
        for (int i=0;i<n;i++)
        {
            conf1[i]=1;
            conf2[i]=1;
        }
        return;
    }


    float *x_c, *x_h;
    float *pex_c, *pex_h;
    float *nex_c, *nex_h;
    float *nccP_c;
    float *nccN_c;

	int *isin_h;
	float *conf1_h;
	float *conf2_h;


    cutilSafeCall(hipMalloc( (void**)&x_c, n*M*sizeof(float)));
    cutilSafeCall(hipMemcpy( x_c, x, n*M*sizeof(float), hipMemcpyHostToDevice ));


    cutilSafeCall(hipMalloc( (void**)&pex_c, N1*M*sizeof(float)));
	cutilSafeCall(hipMemcpy( pex_c, pex,N1*M*sizeof(float), hipMemcpyHostToDevice ));
	
    cutilSafeCall(hipMalloc( (void**)&nex_c, N2*M*sizeof(float)));
    cutilSafeCall(hipMemcpy( nex_c, nex,N2*M*sizeof(float), hipMemcpyHostToDevice ));
	

    cutilSafeCall(hipMalloc( (void**)&nccP_c, n*N1* sizeof(float) ));
    cutilSafeCall(hipMalloc( (void**)&nccN_c, n*N2* sizeof(float) ));   
	
    cutilCheckMsg("Error in ccorr_normed Kernal before pex...");
    int block_size = 128;

    dim3 threads(1,block_size); 
	dim3 threads_new(1,1024); 
   
    dim3 grid1(1/ threads_new.x,n*N1);
    ccorr_normed_nn<<< grid1, threads_new>>>(pex_c,x_c,nccP_c,M,N1,n); //,0,c_stream 
    cutilCheckMsg("Error in ccorr_normed Kernal after pex...");
    hipDeviceSynchronize();
		
    dim3 grid2(1/ threads_new.x,n*N2);
    ccorr_normed_nn<<< grid2, threads_new>>>(nex_c,x_c,nccN_c,M,N2,n);
    cutilCheckMsg("Error in ccorr_normed Kernal after nex...");
    hipDeviceSynchronize();		

    float *maxP_c;
    float *maxN_c;
    float *maxPP_c;
    int *iP_c;

    dim3 grid3(1/ threads.x,(N1+block_size-1)/block_size);
    dim3 grid4(1/ threads.x,(N2+block_size-1)/block_size);

    cutilSafeCall(hipMalloc((void**)&maxP_c, n*sizeof(float)));
    cutilSafeCall(hipMalloc( (void**)&maxPP_c, n*sizeof(float)));
    cutilSafeCall(hipMalloc( (void**)&iP_c, n*sizeof(int)));
    cutilSafeCall(hipMalloc((void**)&maxN_c, n*sizeof(float)));
   
    for(int i=0;i<n;i++)
	{
		get_maxP_maxPP<<<grid3,threads >>>(nccP_c+i*N1,N1,maxP_c+i,maxPP_c+i,iP_c+i);
		hipDeviceSynchronize();
	}

    for(int i=0;i<n;i++)
	{
		get_maxN<<<grid4,threads>>>(nccN_c+i*N2,N2,maxN_c+i);
	    hipDeviceSynchronize();		
	}   


    dim3 grid5(1/ threads.x,(n+block_size-1)/block_size);
    int* isIn_c;
    float* conf1_c;
    float* conf2_c;
    cutilSafeCall(hipMalloc( (void**)&isIn_c, 3*n*sizeof(int)));
    cutilSafeCall(hipMalloc( (void**)&conf1_c, n*sizeof(float)));
    cutilSafeCall(hipMalloc( (void**)&conf2_c, n*sizeof(float)));

    calc_ConfIsin<<<grid5,threads>>>(n,maxP_c,iP_c,maxPP_c,maxN_c,isIn_c,conf1_c,conf2_c);
    hipDeviceSynchronize();
	

    cutilSafeCall(hipMemcpy( isin,isIn_c,3*n*sizeof(int),  hipMemcpyDeviceToHost ));
    cutilSafeCall(hipMemcpy( conf1,conf1_c,n*sizeof(float),  hipMemcpyDeviceToHost ));
    cutilSafeCall(hipMemcpy( conf2,conf2_c,n*sizeof(float),  hipMemcpyDeviceToHost ));


    hipFree(pex_c);
    hipFree(nex_c);
    hipFree(x_c);
   
    hipFree(isIn_c);
    hipFree(conf1_c);
    hipFree(conf2_c);

    hipFree(maxP_c);
    hipFree(iP_c);
    hipFree(maxPP_c);
    hipFree(maxN_c);

    hipFree(nccP_c);
    hipFree(nccN_c);

}
int** CudaFunctions::calcntuples(int x1[],int x2[],int num_x1_col,int num_x2_col)
{
	int *cuda_x1;
    int *cuda_x2;
    int *cuda_x3;
	
	
	int* resp =(int*)malloc(num_x1_col * num_x2_col * 2*sizeof(int));
	
    
	hipMalloc( (void**)&cuda_x1, num_x1_col*sizeof(int));
    hipMemcpy( cuda_x1, x1,num_x1_col*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc( (void**)&cuda_x2, num_x2_col*sizeof(int));
    hipMemcpy( cuda_x2, x2,num_x2_col*sizeof(int), hipMemcpyHostToDevice);
    
	hipMalloc( (void**)&cuda_x3, num_x1_col * num_x2_col * 2*sizeof(int) );
   

    int block_size = 256 ;
    dim3 threads(block_size,1);//16*1 threads per block
    dim3 grid((num_x1_col*num_x2_col+block_size-1)/ threads.x,1/ threads.y);//no of thread blocks in x direction and y direction [x,y]=rows,cols
   
    calc_grid_kernel<<< grid, threads>>>(cuda_x1,cuda_x2,cuda_x3,num_x1_col,num_x2_col);
	hipDeviceSynchronize();
	  
    hipMemcpy( resp, cuda_x3,num_x1_col * num_x2_col *2* sizeof(int), hipMemcpyDeviceToHost); 
	
	int** out=(int**)malloc(2*sizeof(int*));
    out[0]=&resp[0];
    out[1]=&resp[num_x1_col * num_x2_col];
	
	 
	hipFree(cuda_x1);
    hipFree(cuda_x2);
	hipFree(cuda_x3);
	hipDeviceSynchronize();
	return out;
}

